#include "hip/hip_runtime.h"
#ifndef NLP_CLASS_CU
#define NLP_CLASS_CU

#include "NLPclass.h"

// constructor
armtd_NLP::armtd_NLP()
{
    checkJointsPosition = new TYPE[NUM_TIME_STEPS * NUM_FACTORS * 3];
    dk_checkJointsPosition = new TYPE[NUM_TIME_STEPS * NUM_FACTORS * 3 * NUM_FACTORS];
}


// destructor
armtd_NLP::~armtd_NLP()
{
    delete[] checkJointsPosition;
    delete[] dk_checkJointsPosition;
    delete[] g_copy;
}


bool armtd_NLP::set_parameters(
    TYPE* q_des_input,
    ConstantAccelerationCurve* desired_trajectory_input,
    PZsparse* joint_position_input,
    Obstacles* obstacles_input
 ) 
 {
    q_des = q_des_input;
    desired_trajectory = desired_trajectory_input;
    joint_position = joint_position_input;
    obstacles = obstacles_input;

    constraint_number = (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles + 
                        NUM_FACTORS * 4;

    g_copy = new Number[constraint_number];

    return true;
}


bool armtd_NLP::get_nlp_info(
   Index&          n,
   Index&          m,
   Index&          nnz_jac_g,
   Index&          nnz_h_lag,
   IndexStyleEnum& index_style
)
{
    // The problem described NUM_FACTORS variables, x[NUM_FACTORS] through x[NUM_FACTORS] for each joint
    n = NUM_FACTORS;

    // number of inequality constraint
    m = constraint_number;

    nnz_jac_g = m * n;

    // use the C style indexing (0-based)
    index_style = TNLP::C_STYLE;

    return true;
}
// [TNLP_get_nlp_info]

// [TNLP_get_bounds_info]
// returns the variable bounds
bool armtd_NLP::get_bounds_info(
   Index   n,
   Number* x_l,
   Number* x_u,
   Index   m,
   Number* g_l,
   Number* g_u
)
{
    // here, the n and m we gave IPOPT in get_nlp_info are passed back to us.
    // If desired, we could assert to make sure they are what we think they are.
    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in get_bounds_info!");
    }
    if(m != constraint_number){
        WARNING_PRINT("*** Error wrong value of m in get_bounds_info!");
    }

    // lower bounds
    for( Index i = 0; i < n; i++ ) {
        x_l[i] = -1.0;
    }

    // upper bounds  
    for( Index i = 0; i < n; i++ ) {
        x_u[i] = 1.0;
    }

    // collision avoidance constraints
    Index offset = 0;
    for( Index i = offset; i < offset + (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles; i++ ) {
        g_l[i] = -1e19;
        g_u[i] = 0;
    }
    offset += (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles;

    // state limit constraints
    //     minimum joint position
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        g_l[i] = state_limits_lb[i - offset] + qe;
        g_u[i] = state_limits_ub[i - offset] - qe;
    }
    offset += NUM_FACTORS;

    //     maximum joint position
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        g_l[i] = state_limits_lb[i - offset] + qe;
        g_u[i] = state_limits_ub[i - offset] - qe;
    }
    offset += NUM_FACTORS;

    //     minimum joint velocity
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        g_l[i] = -speed_limits[i - offset] + qde;
        g_u[i] = speed_limits[i - offset] - qde;
    }
    offset += NUM_FACTORS;

    //     maximum joint velocity
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        g_l[i] = -speed_limits[i - offset] + qde;
        g_u[i] = speed_limits[i - offset] - qde;
    }

    return true;
}
// [TNLP_get_bounds_info]

// [TNLP_get_starting_point]
// returns the initial point for the problem
bool armtd_NLP::get_starting_point(
    Index   n,
    bool    init_x,
    Number* x,
    bool    init_z,
    Number* z_L,
    Number* z_U,
    Index   m,
    bool    init_lambda,
    Number* lambda
)
{
    // Here, we assume we only have starting values for x, if you code
    // your own NLP, you can provide starting values for the dual variables
    // if you wish
    if(init_x == false || init_z == true || init_lambda == true){
        WARNING_PRINT("*** Error wrong value of init in get_starting_point!");
    }

    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in get_starting_point!");
    }

    for( Index i = 0; i < n; i++ ) {
        // initialize to zero
        // x[i] = 0.0;

        // try to avoid local minimum
        x[i] = min(max((q_des[i] - desired_trajectory->q0[i]) / desired_trajectory->k_range[i], -0.5), 0.5);
    }

    return true;
}
// [TNLP_get_starting_point]

// [TNLP_eval_f]
// returns the value of the objective function
bool armtd_NLP::eval_f(
   Index         n,
   const Number* x,
   bool          new_x,
   Number&       obj_value
)
{
    if(n != NUM_FACTORS){
       WARNING_PRINT("*** Error wrong value of n in eval_f!");
    }

    // obj_value = sum((q_plan - q_des).^2);
    obj_value = 0; 
    for(Index i = 0; i < n; i++){
        obj_value += pow(desired_trajectory->q0[i] + desired_trajectory->qd0[i] * 0.5 + desired_trajectory->k_range[i] * x[i] * 0.125 - q_des[i], 2);
    }

    return true;
}
// [TNLP_eval_f]

// [TNLP_eval_grad_f]
// return the gradient of the objective function grad_{x} f(x)
bool armtd_NLP::eval_grad_f(
   Index         n,
   const Number* x,
   bool          new_x,
   Number*       grad_f
)
{
    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in eval_grad_f!");
    }

    for(Index i = 0; i < n; i++){
        grad_f[i] = 2 * (desired_trajectory->q0[i] + desired_trajectory->qd0[i] * 0.5 + desired_trajectory->k_range[i] * x[i] * 0.125 - q_des[i]) * desired_trajectory->k_range[i] * 0.125;
    }

    return true;
}
// [TNLP_eval_grad_f]

// [TNLP_eval_g]
// return the value of the constraints: g(x)
bool armtd_NLP::eval_g(
   Index         n,
   const Number* x,
   bool          new_x,
   Index         m,
   Number*       g
)
{
    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in eval_g!");
    }
    if(m != constraint_number){
        WARNING_PRINT("*** Error wrong value of m in eval_g!");
    }

    Index i;
    #pragma omp parallel for private(i) schedule(static, NUM_TIME_STEPS * NUM_FACTORS / NUM_THREADS)
    for(i = 0; i < NUM_TIME_STEPS * NUM_FACTORS; i++) {
        checkJointsPosition[i * 3    ] = getCenter(joint_position[i * 3    ].slice(x));
        checkJointsPosition[i * 3 + 1] = getCenter(joint_position[i * 3 + 1].slice(x));
        checkJointsPosition[i * 3 + 2] = getCenter(joint_position[i * 3 + 2].slice(x));
    }

    // Part 3. check collision between joint position reachable set and obstacles (in gpu)
    obstacles->linkFRSConstraints(checkJointsPosition, nullptr, g, nullptr);

    // Part 4. (position & velocity) state limit constraints
    desired_trajectory->returnJointPositionExtremum(g + (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles, x);
    desired_trajectory->returnJointVelocityExtremum(g + (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles + NUM_FACTORS * 2, x);

    return true;
}
// [TNLP_eval_g]


// [TNLP_eval_jac_g]
// return the structure or values of the Jacobian
bool armtd_NLP::eval_jac_g(
   Index         n,
   const Number* x,
   bool          new_x,
   Index         m,
   Index         nele_jac,
   Index*        iRow,
   Index*        jCol,
   Number*       values
)
{
    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in eval_g!");
    }
    if(m != constraint_number){
        WARNING_PRINT("*** Error wrong value of m in eval_g!");
    }
        
    if( values == NULL ) {
       // return the structure of the Jacobian
       // this particular Jacobian is dense
        for(Index i = 0; i < m; i++){
            for(Index j = 0; j < n; j++){
                iRow[i * n + j] = i;
                jCol[i * n + j] = j;
            }
        }
    }
    else {
        Index i;
        #pragma omp parallel for private(i) schedule(static, NUM_TIME_STEPS * NUM_FACTORS / NUM_THREADS)
        for(i = 0; i < NUM_TIME_STEPS * NUM_FACTORS; i++) {
            checkJointsPosition[i * 3    ] = getCenter(joint_position[i * 3    ].slice(x));
            checkJointsPosition[i * 3 + 1] = getCenter(joint_position[i * 3 + 1].slice(x));
            checkJointsPosition[i * 3 + 2] = getCenter(joint_position[i * 3 + 2].slice(x));
            joint_position[i * 3    ].slice(dk_checkJointsPosition + (i * 3    ) * NUM_FACTORS, x);
            joint_position[i * 3 + 1].slice(dk_checkJointsPosition + (i * 3 + 1) * NUM_FACTORS, x);
            joint_position[i * 3 + 2].slice(dk_checkJointsPosition + (i * 3 + 2) * NUM_FACTORS, x);
        }

        // Part 3. check collision between joint position reachable set and obstacles (in gpu)
        obstacles->linkFRSConstraints(checkJointsPosition, dk_checkJointsPosition, nullptr, values);

        // Part 4. (position & velocity) state limit constraints
        desired_trajectory->returnJointPositionExtremumGradient(values + ((NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles) * NUM_FACTORS, x);
        desired_trajectory->returnJointVelocityExtremumGradient(values + ((NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles + NUM_FACTORS * 2) * NUM_FACTORS, x);
    }

    return true;
}
// [TNLP_eval_jac_g]


// [TNLP_eval_h]
//return the structure or values of the Hessian
bool armtd_NLP::eval_h(
   Index         n,
   const Number* x,
   bool          new_x,
   Number        obj_factor,
   Index         m,
   const Number* lambda,
   bool          new_lambda,
   Index         nele_hess,
   Index*        iRow,
   Index*        jCol,
   Number*       values
)
{
    return false;
}
// [TNLP_eval_h]


// [TNLP_finalize_solution]
void armtd_NLP::finalize_solution(
    SolverReturn               status,
    Index                      n,
    const Number*              x,
    const Number*              z_L,
    const Number*              z_U,
    Index                      m,
    const Number*              g,
    const Number*              lambda,
    Number                     obj_value,
    const IpoptData*           ip_data,
    IpoptCalculatedQuantities* ip_cq
)
{
    // here is where we would store the solution to variables, or write to a file, etc
    // so we could use the solution.

    // store the solution
    for( Index i = 0; i < n; i++ ) {
        solution[i] = (TYPE)x[i];
    }

    // check constraint violation manually for Maximum_CpuTime_Exceeded case
    eval_g(n, x, true, m, g_copy);

    feasible = true;

    // collision avoidance constraints
    Index offset = 0;
    for( Index i = 0; i < NUM_FACTORS - 1; i++ ) {
        for( Index j = 0; j < NUM_TIME_STEPS; j++ ) {
            for( Index h = 0; h < obstacles->num_obstacles; h++ ) {
                if (g_copy[(i * NUM_TIME_STEPS + j) * obstacles->num_obstacles + h + offset] > COLLISION_AVOIDANCE_CONSTRAINT_VIOLATION_THRESHOLD) {
                    feasible = false;
                    cout << "        CUDA & C++: Ipopt: Collision between link " << i + 1 << " and obstacle " << h << " at time interval " << j << "!\n";
                    cout << "                        value: " << g_copy[(i * NUM_TIME_STEPS + j) * obstacles->num_obstacles + h + offset] << "\n";
                    return;
                }
            }
        }
    }
    offset += (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles;

    // state limit constraints
    //     minimum joint position
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        if (g_copy[i] < state_limits_lb[i - offset] + qe || g_copy[i] > state_limits_ub[i - offset] - qe) {
            feasible = false;
            cout << "        CUDA & C++: Ipopt: joint " << i - offset << " exceeds position limit when it reaches minimum!\n";
            cout << "                        value: " << g_copy[i] << "\n";
            cout << "                        range: [ " << state_limits_lb[i - offset] + qe << ", "
                                                        << state_limits_ub[i - offset] - qe << " ]\n";
            return;
        }
    }
    offset += NUM_FACTORS;

    //     maximum joint position
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        if (g_copy[i] < state_limits_lb[i - offset] + qe || g_copy[i] > state_limits_ub[i - offset] - qe) {
            feasible = false;
            cout << "        CUDA & C++: Ipopt: joint " << i - offset << " exceeds position limit when it reaches maximum!\n";
            cout << "                        value: " << g_copy[i] << "\n";
            cout << "                        range: [ " << state_limits_lb[i - offset] + qe << ", "
                                                        << state_limits_ub[i - offset] - qe << " ]\n";
            return;
        }
    }
    offset += NUM_FACTORS;

    //     minimum joint velocity
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        if (g_copy[i] < -speed_limits[i - offset] + qde || g_copy[i] > speed_limits[i - offset] - qde) {
            feasible = false;
            cout << "        CUDA & C++: Ipopt: joint " << i - offset << " exceeds velocity limit when it reaches minimum!\n";
            cout << "                        value: " << g_copy[i] << "\n";
            cout << "                        range: [ " << -speed_limits[i - offset] + qde << ", "
                                                        << speed_limits[i - offset] - qde << " ]\n";
            return;
        }
    }
    offset += NUM_FACTORS;

    //     maximum joint velocity
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        if (g_copy[i] < -speed_limits[i - offset] + qde || g_copy[i] > speed_limits[i - offset] - qde) {
            feasible = false;
            cout << "        CUDA & C++: Ipopt: joint " << i - offset << " exceeds velocity limit when it reaches maximum!\n";
            cout << "                        value: " << g_copy[i] << "\n";
            cout << "                        range: [ " << -speed_limits[i - offset] + qde << ", "
                                                        << speed_limits[i - offset] - qde << " ]\n";
            return;
        }
    }
}
// [TNLP_finalize_solution]


#endif
