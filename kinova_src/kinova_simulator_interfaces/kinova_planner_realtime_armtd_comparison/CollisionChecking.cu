#include "hip/hip_runtime.h"
#ifndef COLLISION_CHECKING_CU
#define COLLISION_CHECKING_CU

#include "CollisionChecking.h"

Obstacles::Obstacles(const TYPE* obstacles_inp, const int num_obstacles_inp) {
    obstacles = obstacles_inp;
    num_obstacles = num_obstacles_inp;

    if (num_obstacles > MAX_OBSTACLE_NUM) {
        WARNING_PRINT("Number of obstacles larger than MAX_OBSTACLE_NUM !\n");
        throw;
    }

    // build pre-defined hyper-plane equations for collision checking between links and obstacles
    if (obstacles != nullptr && num_obstacles > 0) {
        hipMalloc((void**)&dev_jointPositionRadius, NUM_FACTORS * 3 * NUM_TIME_STEPS * sizeof(TYPE));
    
        hipMalloc((void**)&dev_A, NUM_FACTORS * NUM_TIME_STEPS * num_obstacles * COMB_NUM * 3 * sizeof(TYPE));
        hipMalloc((void**)&dev_d, NUM_FACTORS * NUM_TIME_STEPS * num_obstacles * COMB_NUM * sizeof(TYPE));
        hipMalloc((void**)&dev_delta, NUM_FACTORS * NUM_TIME_STEPS * num_obstacles * COMB_NUM * sizeof(TYPE));

        // initialize constant memory
        // obstacle data
        hipMemcpyToSymbol(HIP_SYMBOL(dev_obstacles), obstacles, num_obstacles * (MAX_OBSTACLE_GENERATOR_NUM + 1) * 3 * sizeof(TYPE));

        // combination index
        unsigned int combA[COMB_NUM], combB[COMB_NUM];
        unsigned int a_id = 0, b_id = 1;
        for (unsigned int i = 0; i < COMB_NUM; i++) {
            combA[i] = a_id;
            combB[i] = b_id;

            if (b_id < BUFFER_OBSTACLE_GENERATOR_NUM - 1) {
                b_id++;
            }
            else {
                a_id++;
                b_id = a_id + 1;
            }
        }

        hipMemcpyToSymbol(HIP_SYMBOL(dev_combA), combA, COMB_NUM * sizeof(unsigned int));
        hipMemcpyToSymbol(HIP_SYMBOL(dev_combB), combB, COMB_NUM * sizeof(unsigned int));

        hipMalloc((void**)&dev_buffered_c, NUM_TIME_STEPS * num_obstacles * 3 * sizeof(TYPE));
        hipMalloc((void**)&dev_buffered_G, NUM_TIME_STEPS * num_obstacles * (MAX_OBSTACLE_GENERATOR_NUM + 3) * 3 * sizeof(TYPE));

        hipMalloc((void**)&dev_checkJointsPosition, NUM_TIME_STEPS * NUM_FACTORS * 3 * sizeof(TYPE));
        hipMalloc((void**)&dev_dk_checkJointsPosition, NUM_TIME_STEPS * NUM_FACTORS * 3 * NUM_FACTORS * sizeof(TYPE));

        hipMalloc((void**)&dev_link_c, NUM_TIME_STEPS * num_obstacles * sizeof(TYPE));
        hipMalloc((void**)&dev_grad_link_c, NUM_TIME_STEPS * num_obstacles * NUM_FACTORS * sizeof(TYPE));
    }
}

Obstacles::~Obstacles() {
    hipFree(dev_jointPositionRadius);

    hipFree(dev_A);
    hipFree(dev_d);
    hipFree(dev_delta);

    hipFree(dev_buffered_c);
    hipFree(dev_buffered_G);

    hipFree(dev_checkJointsPosition);
    hipFree(dev_dk_checkJointsPosition);

    hipFree(dev_link_c);
    hipFree(dev_grad_link_c);
}

void Obstacles::initializeHyperPlane(const TYPE* jointPositionRadius) {
    if (num_obstacles == 0) return;

    hipMemcpy(dev_jointPositionRadius, jointPositionRadius, NUM_TIME_STEPS * NUM_FACTORS * 3 * sizeof(TYPE), hipMemcpyHostToDevice);

    for (unsigned int i = 0; i < NUM_FACTORS; i++) {
        hipMemset(dev_buffered_G, 0, NUM_TIME_STEPS * num_obstacles * (MAX_OBSTACLE_GENERATOR_NUM + 3) * 3 * sizeof(TYPE));

        dim3 block1(3, num_obstacles);
        bufferObstaclesKernel << < NUM_TIME_STEPS, block1 >> > (dev_jointPositionRadius, link_radius[i][0], link_radius[i][1], link_radius[i][2], dev_buffered_c, dev_buffered_G, i);
        
        dim3 grid1(NUM_TIME_STEPS, num_obstacles);
        polytope_PH << < grid1, COMB_NUM >> > (dev_buffered_c, dev_buffered_G, dev_A, dev_d, dev_delta, i);
    }

    hipDeviceSynchronize();
}

void Obstacles::linkFRSConstraints(TYPE* checkJointsPosition,
                                   TYPE* dk_checkJointsPosition,
                                   TYPE* link_c,
                                   TYPE* grad_link_c) {
    if (num_obstacles == 0) return;
                                    
    bool ifComputeGradient = true;

    hipMemcpy(dev_checkJointsPosition, checkJointsPosition, NUM_TIME_STEPS * NUM_FACTORS * 3 * sizeof(TYPE), hipMemcpyHostToDevice);

    if (dk_checkJointsPosition == nullptr || grad_link_c == nullptr) {
        ifComputeGradient = false;
    }
    else {
        hipMemcpy(dev_dk_checkJointsPosition, dk_checkJointsPosition, NUM_TIME_STEPS * NUM_FACTORS * 3 * NUM_FACTORS * sizeof(TYPE), hipMemcpyHostToDevice);
    }

	// ignore the first link (base -> first joint) collision checking
    for (unsigned int i = 1; i < NUM_FACTORS; i++) {
        dim3 grid1(NUM_TIME_STEPS, num_obstacles);
        
        if (ifComputeGradient) {
            checkCollisionKernel << < grid1, COMB_NUM >> > (dev_A, dev_d, dev_delta,
                                                            dev_checkJointsPosition,
                                                            dev_dk_checkJointsPosition,
                                                            i, 
                                                            dev_link_c, dev_grad_link_c);
        }
        else {
            checkCollisionKernel << < grid1, COMB_NUM >> > (dev_A, dev_d, dev_delta,
                                                            dev_checkJointsPosition,
                                                            nullptr,
                                                            i, 
                                                            dev_link_c, dev_grad_link_c);
        }
            
        if (link_c != nullptr) {
            hipMemcpy(link_c + (i - 1) * NUM_TIME_STEPS * num_obstacles, dev_link_c, NUM_TIME_STEPS * num_obstacles * sizeof(TYPE), hipMemcpyDeviceToHost);
        }
        if (grad_link_c != nullptr) {
            hipMemcpy(grad_link_c + (i - 1) * NUM_TIME_STEPS * num_obstacles * NUM_FACTORS, dev_grad_link_c, NUM_TIME_STEPS * num_obstacles * NUM_FACTORS * sizeof(TYPE), hipMemcpyDeviceToHost);
        }
    }
}

__global__ void bufferObstaclesKernel(TYPE* jointPositionRadius, TYPE link_radius_x, TYPE link_radius_y, TYPE link_radius_z, TYPE* buffered_c, TYPE* buffered_G, int link_id) {
	unsigned int time_id = blockIdx.x;
	unsigned int obs_id = threadIdx.y;
	unsigned int p_id = threadIdx.x;
	unsigned int num_obstacles = blockDim.y;

	__shared__ TYPE Grest[3];

	if (obs_id == 0) {
		if (p_id == 0) {
			Grest[0] = jointPositionRadius[(time_id * NUM_FACTORS + link_id) * 3    ] + link_radius_x;
		}
		else if (p_id == 1) {
			Grest[1] = jointPositionRadius[(time_id * NUM_FACTORS + link_id) * 3 + 1] + link_radius_y;
		}
		else {
			Grest[2] = jointPositionRadius[(time_id * NUM_FACTORS + link_id) * 3 + 2] + link_radius_z;
		}
	}

	__syncthreads();

	// copy obstacle center to buffered obstacle center
	buffered_c[(time_id * num_obstacles + obs_id) * 3 + p_id] = dev_obstacles[obs_id * (MAX_OBSTACLE_GENERATOR_NUM + 1) * 3 + p_id];

	// copy obstacle generator to buffered obstacle generator
	for (unsigned int i = 0; i < MAX_OBSTACLE_GENERATOR_NUM; i++) {
		buffered_G[((time_id * num_obstacles + obs_id) * (MAX_OBSTACLE_GENERATOR_NUM + 3) + i) * 3 + p_id] = dev_obstacles[(obs_id * (MAX_OBSTACLE_GENERATOR_NUM + 1) + i + 1) * 3 + p_id];
	}

	// copy joint position error to buffered obstacle generator
	buffered_G[((time_id * num_obstacles + obs_id) * (MAX_OBSTACLE_GENERATOR_NUM + 3) + p_id + MAX_OBSTACLE_GENERATOR_NUM) * 3 + p_id] = Grest[p_id];
}

__global__ void polytope_PH(TYPE* buffered_c, TYPE* buffered_G, TYPE* A, TYPE* d, TYPE* delta, unsigned int link_id) {
	unsigned int time_id = blockIdx.x;
	unsigned int obs_id = blockIdx.y;
	unsigned int num_obstacles = gridDim.y;
	unsigned int p_id = threadIdx.x;

	unsigned int a_id = dev_combA[p_id];
	unsigned int b_id = dev_combB[p_id];

	__shared__ TYPE C[COMB_NUM][3];
	__shared__ TYPE G[3][MAX_OBSTACLE_GENERATOR_NUM + 3];
	__shared__ TYPE c[3];

	if (p_id < MAX_OBSTACLE_GENERATOR_NUM + 3) {
		G[0][p_id] = buffered_G[((time_id * num_obstacles + obs_id) * (MAX_OBSTACLE_GENERATOR_NUM + 3) + p_id) * 3    ];
		G[1][p_id] = buffered_G[((time_id * num_obstacles + obs_id) * (MAX_OBSTACLE_GENERATOR_NUM + 3) + p_id) * 3 + 1];
		G[2][p_id] = buffered_G[((time_id * num_obstacles + obs_id) * (MAX_OBSTACLE_GENERATOR_NUM + 3) + p_id) * 3 + 2];
	}

	if (p_id < 3) {
		c[p_id] = buffered_c[(time_id * num_obstacles + obs_id) * 3 + p_id];
	}

	__syncthreads();

	TYPE a_x = G[0][a_id];
	TYPE a_y = G[1][a_id];
	TYPE a_z = G[2][a_id];

	TYPE b_x = G[0][b_id];
	TYPE b_y = G[1][b_id];
	TYPE b_z = G[2][b_id];

	TYPE cross_x = a_y * b_z - a_z * b_y;
	TYPE cross_y = a_z * b_x - a_x * b_z;
	TYPE cross_z = a_x * b_y - a_y * b_x;

	TYPE norm_cross = sqrt(cross_x * cross_x + cross_y * cross_y + cross_z * cross_z);

	if (norm_cross > 0) {
		C[p_id][0] = cross_x / norm_cross;
		C[p_id][1] = cross_y / norm_cross;
		C[p_id][2] = cross_z / norm_cross;
	}
	else {
		C[p_id][0] = 0.0;
		C[p_id][1] = 0.0;
		C[p_id][2] = 0.0;
	}

	// A = C
	A[(((time_id * NUM_FACTORS + link_id) * num_obstacles + obs_id) * COMB_NUM + p_id) * 3    ] = C[p_id][0];
	A[(((time_id * NUM_FACTORS + link_id) * num_obstacles + obs_id) * COMB_NUM + p_id) * 3 + 1] = C[p_id][1];
	A[(((time_id * NUM_FACTORS + link_id) * num_obstacles + obs_id) * COMB_NUM + p_id) * 3 + 2] = C[p_id][2];

	// d = C * c
	d[((time_id * NUM_FACTORS + link_id) * num_obstacles + obs_id) * COMB_NUM + p_id] = C[p_id][0] * c[0] + C[p_id][1] * c[1] + C[p_id][2] * c[2];

	// delta = sum(abs(C * G))
	TYPE delta_res = 0.0;

	for (unsigned int j = 0; j < MAX_OBSTACLE_GENERATOR_NUM + 3; j++) {
		delta_res += fabs(C[p_id][0] * G[0][j] + C[p_id][1] * G[1][j] + C[p_id][2] * G[2][j]);
	}

	delta[((time_id * NUM_FACTORS + link_id) * num_obstacles + obs_id) * COMB_NUM + p_id] = delta_res;
}

__global__ void checkCollisionKernel(TYPE* A, TYPE* d, TYPE* delta, 
									 TYPE* checkJointsPosition,
									 TYPE* dk_checkJointsPosition,
									 int link_id, 
                                     TYPE* link_c, TYPE* grad_link_c) {
	unsigned int time_id = blockIdx.x;
	unsigned int obs_id = blockIdx.y;
	unsigned int num_obstacles = gridDim.y;
	unsigned int p_id = threadIdx.x;

	__shared__ TYPE lc1[3];
	__shared__ TYPE lc2[3];
	__shared__ TYPE d_lc1[3][NUM_FACTORS];
	__shared__ TYPE d_lc2[3][NUM_FACTORS];

	__shared__ TYPE C[COMB_NUM][3];

	__shared__ TYPE begin_dot[COMB_NUM];
	__shared__ TYPE end_dot[COMB_NUM];
	__shared__ TYPE delta_v[COMB_NUM];
	__shared__ TYPE lambda_lb[COMB_NUM];
	__shared__ TYPE lambda_ub[COMB_NUM];
	__shared__ TYPE LHS[COMB_NUM];

	if (p_id == 0) {
        if (link_id > 0) {
            lc1[0] = checkJointsPosition[(time_id * NUM_FACTORS + link_id - 1) * 3    ];
            lc1[1] = checkJointsPosition[(time_id * NUM_FACTORS + link_id - 1) * 3 + 1];
            lc1[2] = checkJointsPosition[(time_id * NUM_FACTORS + link_id - 1) * 3 + 2];
        }
        else {
            lc1[0] = 0; // base position x
            lc1[1] = 0; // base position y
            lc1[2] = 0; // base position z
        }
		lc2[0] = checkJointsPosition[(time_id * NUM_FACTORS + link_id) * 3    ];
		lc2[1] = checkJointsPosition[(time_id * NUM_FACTORS + link_id) * 3 + 1];
		lc2[2] = checkJointsPosition[(time_id * NUM_FACTORS + link_id) * 3 + 2];
	}
	
	if (p_id < NUM_FACTORS && dk_checkJointsPosition != nullptr) { // Assume that COMB_NUM is definitely larger than NUM_FACTORS
        if (link_id > 0) {
            d_lc1[0][p_id] = dk_checkJointsPosition[((time_id * NUM_FACTORS + link_id - 1) * 3    ) * NUM_FACTORS + p_id];
            d_lc1[1][p_id] = dk_checkJointsPosition[((time_id * NUM_FACTORS + link_id - 1) * 3 + 1) * NUM_FACTORS + p_id];
            d_lc1[2][p_id] = dk_checkJointsPosition[((time_id * NUM_FACTORS + link_id - 1) * 3 + 2) * NUM_FACTORS + p_id];
        }
        else {
            d_lc1[0][p_id] = 0;
            d_lc1[1][p_id] = 0;
            d_lc1[2][p_id] = 0;
        }
		d_lc2[0][p_id] = dk_checkJointsPosition[((time_id * NUM_FACTORS + link_id) * 3    ) * NUM_FACTORS + p_id];
		d_lc2[1][p_id] = dk_checkJointsPosition[((time_id * NUM_FACTORS + link_id) * 3 + 1) * NUM_FACTORS + p_id];
		d_lc2[2][p_id] = dk_checkJointsPosition[((time_id * NUM_FACTORS + link_id) * 3 + 2) * NUM_FACTORS + p_id];
	}

	lambda_lb[p_id] = -100000000;
	lambda_ub[p_id] = 100000000;
	LHS[p_id] = 0;

	__syncthreads();

	TYPE C1 = A[(((time_id * NUM_FACTORS + link_id) * num_obstacles + obs_id) * COMB_NUM + p_id) * 3    ];
	TYPE C2 = A[(((time_id * NUM_FACTORS + link_id) * num_obstacles + obs_id) * COMB_NUM + p_id) * 3 + 1];
	TYPE C3 = A[(((time_id * NUM_FACTORS + link_id) * num_obstacles + obs_id) * COMB_NUM + p_id) * 3 + 2];
	C[p_id][0] = C1;
	C[p_id][1] = C2;
	C[p_id][2] = C3;

	if (C1 != 0 || C2 != 0 || C3 != 0) {
		TYPE d_v = d[((time_id * NUM_FACTORS + link_id) * num_obstacles + obs_id) * COMB_NUM + p_id];
		delta_v[p_id] = delta[((time_id * NUM_FACTORS + link_id) * num_obstacles + obs_id) * COMB_NUM + p_id];
		begin_dot[p_id] = C1 * lc1[0] + C2 * lc1[1] + C3 * lc1[2] - d_v;
		end_dot[p_id] = C1 * lc2[0] + C2 * lc2[1] + C3 * lc2[2] - d_v;
		LHS[p_id] = begin_dot[p_id] - end_dot[p_id];

		if (LHS[p_id] > 0) {
			lambda_ub[p_id] = (delta_v[p_id] - end_dot[p_id]) / LHS[p_id];
            lambda_lb[p_id] = (-delta_v[p_id] - end_dot[p_id]) / LHS[p_id];
		}
		else {
			lambda_ub[p_id] = (-delta_v[p_id] - end_dot[p_id]) / LHS[p_id];
            lambda_lb[p_id] = (delta_v[p_id] - end_dot[p_id]) / LHS[p_id];
		}
	}

	__syncthreads();
	
	if (p_id == 0) {
		unsigned int lb_max_id = 0;
		unsigned int ub_min_id = 0;
		for (unsigned int i = 1; i < COMB_NUM; i++) {
			if (lambda_lb[lb_max_id] < lambda_lb[i]) {
				lb_max_id = i;
			}
			if (lambda_ub[i] < lambda_ub[ub_min_id]) {
				ub_min_id = i;
			}
		}
		
		const TYPE& lambda_lb_max = lambda_lb[lb_max_id];
		const TYPE& LHS_lb = LHS[lb_max_id];
		const TYPE begin_dot_lb = begin_dot[lb_max_id];
		const TYPE end_dot_lb = end_dot[lb_max_id];
		const TYPE& delta_lb = delta_v[lb_max_id];

		const TYPE& lambda_ub_min = lambda_ub[ub_min_id];
		const TYPE& LHS_ub = LHS[ub_min_id];
		// const TYPE begin_dot_ub = begin_dot[ub_min_id];
		const TYPE end_dot_ub = end_dot[ub_min_id];
		const TYPE& delta_ub = delta_v[ub_min_id];
		
		TYPE begin_dot_lb_derivative[NUM_FACTORS];
		TYPE end_dot_lb_derivative[NUM_FACTORS];
		TYPE begin_dot_ub_derivative[NUM_FACTORS];
		TYPE end_dot_ub_derivative[NUM_FACTORS];

		if (dk_checkJointsPosition != nullptr) {
			for (unsigned int i = 0; i < NUM_FACTORS; i++) {
				begin_dot_lb_derivative[i] = C[lb_max_id][0] * d_lc1[0][i] + C[lb_max_id][1] * d_lc1[1][i] + C[lb_max_id][2] * d_lc1[2][i];
				end_dot_lb_derivative[i] = C[lb_max_id][0] * d_lc2[0][i] + C[lb_max_id][1] * d_lc2[1][i] + C[lb_max_id][2] * d_lc2[2][i];
				begin_dot_ub_derivative[i] = C[ub_min_id][0] * d_lc1[0][i] + C[ub_min_id][1] * d_lc1[1][i] + C[ub_min_id][2] * d_lc1[2][i];
				end_dot_ub_derivative[i] = C[ub_min_id][0] * d_lc2[0][i] + C[ub_min_id][1] * d_lc2[1][i] + C[ub_min_id][2] * d_lc2[2][i];
			}
		}

		if (1 < lambda_lb_max) {
			if (LHS_lb > 0) {
				link_c[time_id * num_obstacles + obs_id] = begin_dot_lb - delta_lb;
				
				for (unsigned int i = 0; i < NUM_FACTORS; i++) {
					grad_link_c[(time_id * num_obstacles + obs_id) * NUM_FACTORS + i] = begin_dot_lb_derivative[i];
				}
			}
			else {
				link_c[time_id * num_obstacles + obs_id] = -begin_dot_lb - delta_lb;

				for (unsigned int i = 0; i < NUM_FACTORS; i++) {
					grad_link_c[(time_id * num_obstacles + obs_id) * NUM_FACTORS + i] = -begin_dot_lb_derivative[i];
				}
			}
		}
		else if (lambda_ub_min < 0) {
			if (LHS_ub > 0) {
				link_c[time_id * num_obstacles + obs_id] = -end_dot_ub + delta_ub;

				for (unsigned int i = 0; i < NUM_FACTORS; i++) {
					grad_link_c[(time_id * num_obstacles + obs_id) * NUM_FACTORS + i] = -end_dot_ub_derivative[i];
				}
			}
			else {
				link_c[time_id * num_obstacles + obs_id] = end_dot_ub + delta_ub;

				for (unsigned int i = 0; i < NUM_FACTORS; i++) {
					grad_link_c[(time_id * num_obstacles + obs_id) * NUM_FACTORS + i] = end_dot_ub_derivative[i];
				}
			}
		}
		else {
			TYPE LHS_ub_derivative[NUM_FACTORS];
			TYPE LHS_lb_derivative[NUM_FACTORS];

			if (dk_checkJointsPosition != nullptr) {
				for (unsigned int i = 0; i < NUM_FACTORS; i++) {
					LHS_ub_derivative[i] = begin_dot_ub_derivative[i] - end_dot_ub_derivative[i];
					LHS_lb_derivative[i] = begin_dot_lb_derivative[i] - end_dot_lb_derivative[i];
				}
			}

			if (LHS_lb > 0) {
				if (LHS_ub > 0) {
					link_c[time_id * num_obstacles + obs_id] = LHS_ub * (end_dot_lb + delta_lb) + LHS_lb * (-end_dot_ub + delta_ub);

					for (unsigned int i = 0; i < NUM_FACTORS; i++) {
						grad_link_c[(time_id * num_obstacles + obs_id) * NUM_FACTORS + i] = 
										(LHS_ub_derivative[i] * (end_dot_lb + delta_lb)) + 
										(LHS_ub * end_dot_lb_derivative[i]) + 
										(LHS_lb_derivative[i] * (-end_dot_ub + delta_ub)) + 
										(LHS_lb * (-end_dot_ub_derivative[i]));
					}
				}
				else {
					link_c[time_id * num_obstacles + obs_id] = -LHS_ub * (end_dot_lb + delta_lb) + LHS_lb * (end_dot_ub + delta_ub);

					for (unsigned int i = 0; i < NUM_FACTORS; i++) {
						grad_link_c[(time_id * num_obstacles + obs_id) * NUM_FACTORS + i] = 
										(-LHS_ub_derivative[i] * (end_dot_lb + delta_lb)) + 
										(-LHS_ub * end_dot_lb_derivative[i]) + 
										(LHS_lb_derivative[i] * (end_dot_ub + delta_ub)) + 
										(LHS_lb * (end_dot_ub_derivative[i]));
					}
				}
			}
			else {
				if (LHS_ub > 0) {
					link_c[time_id * num_obstacles + obs_id] = LHS_ub * (-end_dot_lb + delta_lb) - LHS_lb * (-end_dot_ub + delta_ub);

					for (unsigned int i = 0; i < NUM_FACTORS; i++) {
						grad_link_c[(time_id * num_obstacles + obs_id) * NUM_FACTORS + i] = 
										(LHS_ub_derivative[i] * (-end_dot_lb + delta_lb)) + 
										(LHS_ub * (-end_dot_lb_derivative[i])) + 
										(-LHS_lb_derivative[i] * (-end_dot_ub + delta_ub)) + 
										(LHS_lb * (end_dot_ub_derivative[i]));
					}
				}
				else {
					link_c[time_id * num_obstacles + obs_id] = -LHS_ub * (-end_dot_lb + delta_lb) - LHS_lb * (end_dot_ub + delta_ub);
				
					for (unsigned int i = 0; i < NUM_FACTORS; i++) {
						grad_link_c[(time_id * num_obstacles + obs_id) * NUM_FACTORS + i] = 
										(-LHS_ub_derivative[i] * (-end_dot_lb + delta_lb)) + 
										(LHS_ub * (end_dot_lb_derivative[i])) + 
										(-LHS_lb_derivative[i] * (end_dot_ub + delta_ub)) + 
										(-LHS_lb * (end_dot_ub_derivative[i]));
					}
				}
			}
		}
	}
}

#endif