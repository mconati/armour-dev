#include "hip/hip_runtime.h"
#include "NLPclass.h"
#include "FastCollisionChecking.h"
#include "ReachsetsPath.h"
#include <cmath>

#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <pybind11/eigen.h>
#include <memory>

namespace py = pybind11;

#define NUM_EDGES 112826403 // 50221615
#define COLLISION_THRESHOLD -0.075 // -0.10
#define EDGE_THRESHOLD 0.45 // 1000

class pzsparse {
    private:

        std::string data_dir;
        const std::string ofile1 = "/armour.out";
        const std::string ofile2 = "/armour_joint_position_center.out";
        const std::string ofile3 = "/armour_joint_position_radius.out";

        // Eigen::VectorXd a ({1.5, 2.5, 3.5});
        // Eigen::MatrixXd b ({1.5, 2.5, 3.5});

        Eigen::VectorXd q0 = Eigen::VectorXd::Zero(NUM_FACTORS);
        Eigen::VectorXd qd0 = Eigen::VectorXd::Zero(NUM_FACTORS);
        Eigen::VectorXd qdd0 = Eigen::VectorXd::Zero(NUM_FACTORS);
        Eigen::VectorXd q_des = Eigen::VectorXd::Zero(NUM_FACTORS);

        // Eigen::VectorXd q0; //q0.setZero(NUM_FACTORS);
        // Eigen::MatrixXd qd0; //qd0.setZero();
        // Eigen::MatrixXd qdd0; //qdd0.setZero();
        // Eigen::MatrixXd q_des; //q_des.setZero();
 
        double t_plan = 0.5*DURATION; // 0.5; // 0.5; // 
        // Kinova Hardware Demo Values: u_s = 0.609382421; surf_rad =  0.058/2;
        double u_s = 0.6;  // 0.396674703; // 0.3358; // 0.609382421; // 0.5; // static coefficient of friction between tray and object
        double surf_rad =  0.058 / 2; // 0.0762; // RADIUS of contact area between tray and object (area assumed to be circular) 
        // Note: might want to change this to be input to the C++ code from matlab?

        int num_obstacles = 0;
        double obstacles[MAX_OBSTACLE_NUM * (MAX_OBSTACLE_GENERATOR_NUM + 1) * 3] = {0.0};
        std::shared_ptr<Obstacles> O_ptr{nullptr};
        std::shared_ptr<SimplifiedObstacles> O_ptr2{nullptr};
        // Obstacles O(obstacles, num_obstacles); 
        // Obstacles O;

        BezierCurve traj;
        PZsparse p[NUM_TIME_STEPS * NUM_FACTORS * 3];
        PZsparse u_nom[NUM_TIME_STEPS * NUM_FACTORS];
        double v_norm[NUM_TIME_STEPS * NUM_FACTORS];
        double jointPositionRadius[NUM_TIME_STEPS * NUM_FACTORS * 3];

        void set_obstacles(py::array_t<double> obstacle_vec){
            auto obstacle_ = obstacle_vec.unchecked<2>();
            num_obstacles = obstacle_.shape(0);
            int row = 0;
            int col = 0;
            int obs_dim = (MAX_OBSTACLE_GENERATOR_NUM + 1) * 3;
            cout << "Obstacles set to:" << endl;
            for (int i = 0; i < num_obstacles * obs_dim; i++) {
                row = i / obs_dim;
                col = i % obs_dim;
                obstacles[i] = obstacle_(row, col);
            }
            for (int i = 0; i <= row; i++){
                cout << "obstacle" << i+1 << ": [ ";
                for (int j = 0; j <= col; j++){
                    cout << obstacles[i*obs_dim + j] << ' ';
                }
                cout << "] \n";
            }
            cout << '\n' << endl;
            std::cout << "allocating obstacles..." << std::endl;
            O_ptr = std::make_unique<Obstacles>();
            O_ptr->initialize(obstacles, num_obstacles);
            // O.initialize(obstacles, num_obstacles);

            O_ptr2 = std::make_unique<SimplifiedObstacles>();
            O_ptr2->initialize(obstacles, num_obstacles);

            std::cout << "Obstacles allocated!" << std::endl;
        }

        void set_goal(Eigen::Ref<Eigen::VectorXd> qdes_vec){
            auto qdes_ = qdes_vec; //.unchecked<1>();
            // cout << "Goal set to: \n [";
            for (int i = 0; i < NUM_FACTORS; i++) {
                q_des[i] = qdes_(i);
                // cout << q_des[i] << ' ';
            }
            // cout << "]" << endl;
        }

        void set_state(Eigen::Ref<Eigen::VectorXd> q0_vec, Eigen::Ref<Eigen::VectorXd> qd0_vec, Eigen::Ref<Eigen::VectorXd> qdd0_vec){
            auto q0_ = q0_vec; //.unchecked<1>();
            auto qd0_ = qd0_vec; //.unchecked<1>();
            auto qdd0_ = qdd0_vec; //.unchecked<1>();
            // cout << "States set to:" << endl;
            // cout << "[q0,     qd0,     qdd0]" << endl;
            for (int i = 0; i < NUM_FACTORS; i++){
                q0[i] = q0_(i);
                qd0[i] = qd0_(i);
                qdd0[i] = qdd0_(i);
                // cout << q0[i] << ' ' << qd0[i] << ' ' << qdd0[i] << endl;
            }
        }

        // void write_reachset(SmartPtr<armtd_NLP> mynlp){
        //     cout << "Saving reachsets..." << endl;
        //     for (int link = 0; link < 7; link++){
        //         std::ofstream file(reachset+std::to_string(link+1)+".txt");
        //         file << std::setprecision(8);
        //         for (int point = 0; point < 8; point++){
        //             int axis1 = static_cast<int>(point / 4);
        //             int axis2 = static_cast<int>(point / 2);
        //             int axis3 = point;
        //             int axis[] = {axis1, axis2, axis3};
        //             for (int i = 0; i < NUM_TIME_STEPS; i++) {
        //                 for (int j = 0; j < 3; j++) {
        //                     file << mynlp->checkJointsPosition[(i * NUM_FACTORS + link) * 3 + j] + (link_radius[link][0]+jointPositionRadius[(i * NUM_FACTORS + link) * 3 + j])*std::pow(-1, axis[j])<< ',';
        //                 }
        //                 file << '\n';
        //             }
        //         }
        //         file.close();
        //     }
        //     // write the origin
        //     std::ofstream file(reachset+"0.txt");
        //     for (int point = 0; point<8; point++){
        //         int axis1 = static_cast<int>(point / 4);
        //         int axis2 = static_cast<int>(point / 2);
        //         int axis3 = point;
        //         int axis[] = {axis1, axis2, axis3};
        //         for (int j = 0; j < 3; j++) {
        //             file << (0.04)*std::pow(-1, axis[j])<< ',';
        //         }
        //         file << '\n';
        //     }
        // }

    public:
    
        pzsparse(py::array_t<double> obs_vec, const std::string &dir)
        : data_dir(dir), num_obstacles(0){
            
            set_obstacles(obs_vec);
        }

        ~pzsparse()
        {
        }

        const int getNumObstacles(){
            return num_obstacles;
        }
        
        py::array_t<double> optimize(Eigen::Ref<Eigen::VectorXd> q0_vec, Eigen::Ref<Eigen::VectorXd> qd0_vec, Eigen::Ref<Eigen::VectorXd> qdd0_vec, 
                                    Eigen::Ref<Eigen::VectorXd> qgoal_vec) {
            

            set_goal(qgoal_vec);
            set_state(q0_vec, qd0_vec, qdd0_vec);      

            auto start1 = std::chrono::high_resolution_clock::now();      

            // Create JRS online
            traj = BezierCurve(q0, qd0, qdd0);
            omp_set_num_threads(NUM_THREADS);
            int openmp_t_ind = 0; // openmp loop index

            try {
                #pragma omp parallel for shared(traj) private(openmp_t_ind) schedule(static, NUM_TIME_STEPS / NUM_THREADS)
                for(openmp_t_ind = 0; openmp_t_ind < NUM_TIME_STEPS; openmp_t_ind++) {
                    traj.makePolyZono(openmp_t_ind);
                }
            }
            catch (int errorCode) {
                WARNING_PRINT("        CUDA & C++: Error creating JRS! Check previous error message!");
                throw;
            }

            // Compute link PZs and nominal torque PZs
            KinematicsDynamics kd(&traj);
            Eigen::Matrix<double, 3, 3 + 3> link_independent_generators[NUM_TIME_STEPS * NUM_JOINTS];

            try {
                #pragma omp parallel for shared(kd, link_independent_generators) private(openmp_t_ind) schedule(static, NUM_TIME_STEPS / NUM_THREADS)
                for(openmp_t_ind = 0; openmp_t_ind < NUM_TIME_STEPS; openmp_t_ind++) {
                    // compute link PZs through forward kinematics
                    kd.fk(openmp_t_ind);

                    // reduce non-only-k-dependent generators so that slice takes less time
                    for (int i = 0; i < NUM_JOINTS; i++) {
                        link_independent_generators[openmp_t_ind * NUM_JOINTS + i] = kd.links(i, openmp_t_ind).reduce_link_PZ();
                    }

                    // compute nominal torque
                    kd.rnea_nominal(openmp_t_ind);

                    // compute interval torque
                    kd.rnea_interval(openmp_t_ind);

                    // compute max disturbance (stored in u_nom_int)
                    for (int i = 0; i < NUM_FACTORS; i++) {
                        kd.u_nom_int(i, openmp_t_ind) = kd.u_nom_int(i, openmp_t_ind) - kd.u_nom(i, openmp_t_ind);
                        // kd.u_nom_int(i, openmp_t_ind) = 0;
                    }

                    // reduce non-only-k-dependent generators so that slice takes less time
                    for (int i = 0; i < NUM_FACTORS; i++) {
                        kd.u_nom(i, openmp_t_ind).reduce();
                    }
                }
            }
            catch (int errorCode) {
                WARNING_PRINT("        CUDA & C++: Error computing link PZs and nominal torque PZs! Check previous error message!");
                throw;
            }

            // Compute robust input bound
            Eigen::MatrixXd torque_radius(NUM_FACTORS, NUM_TIME_STEPS);
            torque_radius.setZero();

            try {
                for(int t_ind = 0; t_ind < NUM_TIME_STEPS; t_ind++) {
                    // (1) add the bound of robust input (||v||)
                    Interval rho_max_temp = Interval(0.0);
                    for (int i = 0; i < NUM_FACTORS; i++) {
                        // compute norm of disturbance
                        MatrixXInt temp = kd.u_nom_int(i, t_ind).toInterval(); // should be a 1-dim Interval
                        rho_max_temp += temp(0) * temp(0);

                        torque_radius(i, t_ind) = alpha * (M_max - M_min) * eps + 0.5 * max(abs(temp(0).lower()), abs(temp(0).upper()));
                        // torque_radius(i, t_ind) = 0;
                    }
                    rho_max_temp = sqrt(rho_max_temp);
                    
                    for (int i = 0; i < NUM_FACTORS; i++) {
                        torque_radius(i, t_ind) += 0.5 * rho_max_temp.upper();
                        // torque_radius(i, t_ind) += 0;
                    }

                    // (2) add the radius of the nominal input PZ (after reducing)
                    for (int i = 0; i < NUM_FACTORS; i++) {
                        torque_radius(i, t_ind) += kd.u_nom(i, t_ind).independent(0);
                        // torque_radius(i, t_ind) += 0;
                    }

                    // (3) add friction
                    for (int i = 0; i < NUM_FACTORS; i++) {
                        torque_radius(i, t_ind) += friction[i];
                        // torque_radius(i, t_ind) += 0;
                    }

                    // so that torque_radius would be the radius of the total control input PZ from now
                }
            }
            catch (int errorCode) {
                WARNING_PRINT("        CUDA & C++: Error computing torque PZs! Check previous error message!");
                throw;
            }

            // Buffer obstacles and initialize collision checking hyperplanes
            try {
                O_ptr->initializeHyperPlane(link_independent_generators);
                // O.initializeHyperPlane(link_independent_generators);
            }
            catch (int errorCode) {
                WARNING_PRINT("        CUDA & C++: Error initializing collision checking hyperplanes! Check previous error message!");
                throw;
            }

            auto stop1 = std::chrono::high_resolution_clock::now();
            auto duration1 = std::chrono::duration_cast<std::chrono::milliseconds>(stop1 - start1);
            cout << "        CUDA & C++: Time taken by generating reachable sets: " << duration1.count() << " milliseconds" << endl;

            auto start2 = std::chrono::high_resolution_clock::now();

            // Solve optimization
            SmartPtr<armtd_NLP> mynlp = new armtd_NLP();
            try {
                mynlp->set_parameters(q_des, t_plan, &traj, &kd, &torque_radius, O_ptr.get(), u_s, surf_rad);
            }
            catch (int errorCode) {
                WARNING_PRINT("        CUDA & C++: Error initializing Ipopt! Check previous error message!");
                throw;
            }

            SmartPtr<IpoptApplication> app = IpoptApplicationFactory();

            app->Options()->SetNumericValue("tol", IPOPT_OPTIMIZATION_TOLERANCE);
            app->Options()->SetNumericValue("max_cpu_time", IPOPT_MAX_CPU_TIME);
            app->Options()->SetIntegerValue("print_level", IPOPT_PRINT_LEVEL);
            app->Options()->SetStringValue("mu_strategy", IPOPT_MU_STRATEGY);
            app->Options()->SetStringValue("linear_solver", IPOPT_LINEAR_SOLVER);
            app->Options()->SetStringValue("hessian_approximation", "limited-memory");

            // For gradient checking
            // app->Options()->SetStringValue("output_file", "ipopt.out");
            // app->Options()->SetStringValue("derivative_test", "first-order");
            // app->Options()->SetNumericValue("derivative_test_perturbation", 1e-8);
            // app->Options()->SetNumericValue("derivative_test_tol", 1e-6);

            // Initialize the IpoptApplication and process the options
            ApplicationReturnStatus status;
            auto outputstream1 = std::ofstream(data_dir + ofile1);
            if (!outputstream1)
            {
                throw std::runtime_error("Open filestream failed!");
            }
            std::cout << "is_open: " << (bool)outputstream1 << std::endl;
            status = app->Initialize();
            if( status != Solve_Succeeded ) {
                WARNING_PRINT("Error during initialization!");
                outputstream1 << -1 << '\n';
                outputstream1.close();
                throw;
            }

            try {
                // Ask Ipopt to solve the problem
                status = app->OptimizeTNLP(mynlp);
            }
            catch (int errorCode) {
                WARNING_PRINT("        CUDA & C++: Error solving optimization problem! Check previous error message!");
                throw;
            }
            
            auto stop2 = std::chrono::high_resolution_clock::now();
            auto duration2 = std::chrono::duration_cast<std::chrono::milliseconds>(stop2 - start2);

            if (status == Maximum_CpuTime_Exceeded) {
                cout << "        CUDA & C++: Ipopt maximum CPU time exceeded!\n";
            }
            
            if (status == Invalid_Option) {
                cout << "        CUDA & C++: Cannot find HSL library! Need to put libcoinhsl.so in proper path!\n";
            }
            else {
                cout << "        CUDA & C++: Time taken by Ipopt: " << duration2.count() << " milliseconds" << endl;
            }

            // set precision to 10 decimal digits
            outputstream1 << std::setprecision(10);

            // output k_opt
            auto k_opt = py::array_t<double>(NUM_FACTORS);
            double *k_opt_ptr = static_cast<double *>(k_opt.request().ptr);

            cout << "cpp planner raw solution: \n [ ";
            if (mynlp->feasible) {
                for (int i = 0; i < NUM_FACTORS; i++) {
                    cout << mynlp->solution[i] << ' ';
                    k_opt_ptr[i] = mynlp->solution[i] * k_range[i];
                }
            }
            else {
                for (int i = 0; i < NUM_FACTORS; i++) {
                    k_opt_ptr[i] = 0;
                }
                cout << "No feasible solution. ";
            }
            cout << "]" << endl;

            // output time cost (in milliseconds) in C++
            outputstream1 << duration1.count() + duration2.count();
            outputstream1.close();

            // output FRS and other information, you can comment them if they are unnecessary
            cout << "saving reach sets ..." << endl;
            std::ofstream outputstream2(data_dir + ofile2);
            outputstream2 << std::setprecision(10);
            for (int i = 0; i < NUM_TIME_STEPS; i++) {
                for (int j = 0; j < NUM_JOINTS; j++) {
                    for (int l = 0; l < 3; l++) {
                        outputstream2 << mynlp->link_sliced_center[i * NUM_JOINTS + j](l) << ' ';
                    }
                    outputstream2 << '\n';
                }
                outputstream2 << '\n';
            }
            outputstream2.close();

            std::ofstream outputstream3(data_dir + ofile3);
            outputstream3 << std::setprecision(10);
            for (int i = 0; i < NUM_TIME_STEPS; i++) {
                for (int j = 0; j < NUM_JOINTS; j++) {
                    for (int k = 0; k < 3; k++) {
                        for (int l = 0; l < 3 + 3; l++) {
                            outputstream3 << link_independent_generators[i * NUM_JOINTS + j](k, l) << ' ';
                        }
                        outputstream3 << '\n';
                    }
                    outputstream3 << '\n';
                }
                outputstream3 << '\n';
            }
            outputstream3.close();

            // std::ofstream outputstream4(outputfilename4);
            // outputstream4 << std::setprecision(10);
            // for (int i = 0; i < NUM_TIME_STEPS; i++) {
            //     for (int j = 0; j < NUM_FACTORS; j++) {
            //         outputstream4 << torque_radius(j, i) << ' '; // this is radius of final control input
            //     }
            //     outputstream4 << '\n';
            // }
            // outputstream4.close();

            // std::ofstream outputstream5(outputfilename5);
            // outputstream5 << std::setprecision(6);
            // for (int i = 0; i < mynlp->constraint_number; i++) {
            //     outputstream5 << mynlp->g_copy[i] << '\n';
            // }
            // outputstream5.close();
            
            return k_opt;
        }

        py::array_t<double> getDesTraj(Eigen::Ref<Eigen::VectorXd> q0, Eigen::Ref<Eigen::VectorXd> qd0, Eigen::Ref<Eigen::VectorXd> qdd0, py::array_t<double> k, double t){
            // return desired trajectory: [q, qd, qdd]
            int size = NUM_FACTORS * 3;
            auto traj_d = py::array_t<double>(size);
            double *traj_d_ptr = static_cast<double *>(traj_d.request().ptr);

            auto q0_ = q0;//.unchecked<1>();
            auto qd0_ = qd0 * DURATION;//.unchecked<1>();
            auto qdd0_ = qdd0 * DURATION * DURATION;//.unchecked<1>();
            t /= DURATION;
            auto k_ = k.unchecked<1>();

            for (int i = 0; i < NUM_FACTORS; i++){
                traj_d_ptr[3*i] = q_des_func(q0_(i), qd0_(i), qdd0_(i), k_(i), t);
                traj_d_ptr[3*i+1] = (qd_des_func(q0_(i), qd0_(i), qdd0_(i), k_(i), t))/DURATION;
                traj_d_ptr[3*i+2] = (qdd_des_func(q0_(i), qd0_(i), qdd0_(i), k_(i), t))/(DURATION*DURATION);
            }
            
            return traj_d;
        }

        // add collision checking function here and below in pybind function
        // take in the obstacles (from the pybind class above?), the precomputed joint locations from the matlab script, and adjacency matrix. 
        // return new adjacency matrix?

        

        // the pybind is initialized in bulletPlanner.py with the initialization of a armour planner

         py::array_t<double> graphCollisionCheck(Eigen::MatrixXd old_adj_matrix, Eigen::MatrixXd joint_positions) {
            /*
            Eigen::Ref<Eigen::MatrixXd>

            This function collision checks nodes in a graph and removes those in collision from the adjacency matrix. A new, collision-free adjacency
            matrix is returned.

            TODO: 
            - replace reading the old adjacency matrix from a file to taking in the adjacency matrix from python. can pass from csr_matrix?
            - can potentially return directly to csr_matrix? and also pass in from csr_matrix?
            */

            std::string output_message = "In C++ graphCollisionCheck";
            std::cout << output_message << std::endl;

            std::string pathname_collision = "/home/baiyuew/ROAHM/planning_wksp/src/kinova_perception/kinova_planning/src/rtd-pybullet/armour-dev/kinova_src/kinova_simulator_interfaces/kinova_planner_realtime/";

            // file names
            // composite set
                // joint_positions_composite.csv
                // adj_matrix_composite_range0p3.txt
                // edges: 28338492
            // uniform nodes hardware only version 2
                // joint_positions_uniform_v2.csv
                // adj_matrix_uniform_hardware_only_v2_range0p3.txt
                // edges: 16140775
            // uniform nodes hardware only version 3
                // joint_positions_uniform_v3.csv
                // adj_matrix_uniform_hardware_only_v3_range0p3.txt
                // edges: 17740918
            // uniform nodes hardware only version 4
                // joint_positions_uniform_v4.csv
                // adj_matrix_uniform_hardware_only_v4_range0p3.txt
                // edges: 17707982
            // uniform nodes hardware only version 4 dense and shifted valid only
                // joint_positions_uniform_dense_validOnly_v4.csv
                // adj_matrix_uniform_hardware_dense_shiftedValidOnly_v4_range0p3.txt
                // edges: 5482178
            // uniform nodes hardware dense shifted and random nodes
                // joint_positions_uniform_hardware_dense_rand.csv
                // adj_matrix_uniform_hardware_dense_rand_range0p3.txt
                // edges: 11930750
            // matlab files
                // joint_positions_uniform.csv
                // adj_matrix_uniform_mult5.txt

            // Hard Coded Files (need to be replaced if graph changes)
            const std::string inputfilename2 = pathname_collision + "joint_positions_uniform.csv";
            const std::string inputfilename3 = pathname_collision + "adj_matrix_uniform_mult5.csv";
            // const std::string outputfilename1 = pathname_collision + "node_feasibility.csv";
            // const std::string outputfilename2 = pathname_collision + "link_c.csv";
            const std::string outputfilename3 = pathname_collision + "collision_free_adj_matrix.csv";

            // O.initialize(obstacles, num_obstacles); // done above

            Eigen::Vector3d link_sliced_center[NUM_NODES_AT_ONE_TIME * NUM_JOINTS];
            Eigen::Matrix<double, 3, LINK_FRS_GENERATOR_NUM> link_independent_generators[NUM_NODES_AT_ONE_TIME * NUM_JOINTS];
            double* link_c = new double[NUM_JOINTS * NUM_NODES_AT_ONE_TIME * num_obstacles];
            bool* node_feasibilities = new bool[NUM_NODES];
            // std::ifstream inputstream2(inputfilename2);
            // std::ofstream outputstream1(outputfilename1);
            // std::ofstream outputstream2(outputfilename2);


            // pre-allocate new adjacency matrix

            // Eigen::MatrixXi new_adj_nodes = Eigen::MatrixXi::Zero(NUM_EDGES,2);
            // Eigen::VectorXd new_edge_distance = Eigen::VectorXd::Zero(NUM_EDGES);

            // Eigen::MatrixXd new_adj_matrix = Eigen::MatrixXd::Zero(NUM_EDGES,3);

            auto new_adj_matrix = py::array_t<double>(NUM_EDGES*3);
            double *new_adj_matrix_ptr = static_cast<double *>(new_adj_matrix.request().ptr);


            auto start1 = std::chrono::high_resolution_clock::now();

            int joint_positions_node_spacing = 9;
            int jpns = joint_positions_node_spacing;
            // reading joint positions
            for (int k = 0; k < NUM_NODES / NUM_NODES_AT_ONE_TIME; k++) {
                // int offset1 = k*NUM_NODES_AT_ONE_TIME-2*k;
                int set_offset = k*NUM_NODES_AT_ONE_TIME;
                for (int i = 0; i < NUM_NODES_AT_ONE_TIME; i++) {
                    Eigen::Vector3d pos1;
                    // inputstream2 >> pos1(0) >> pos1(1) >> pos1(2);
                    // extract the base joint position (every 8th in joint positions?)
                    // pos1(0) = joint_positions(i*jpns+offset1,0);
                    // pos1(1) = joint_positions(i*jpns+offset1,1);
                    // pos1(2) = joint_positions(i*jpns+offset1,2);
                    int first_row = (i+set_offset)*jpns;
                    pos1(0) = joint_positions(first_row,0);
                    pos1(1) = joint_positions(first_row,1);
                    pos1(2) = joint_positions(first_row,2);
                    Eigen::Vector3d pos2;
                    for (int j = 0; j < NUM_JOINTS; j++) {
                        // inputstream2 >> pos2(0) >> pos2(1) >> pos2(2);
                        // pos2(0) = joint_positions(i*jpns+j+offset1,0);
                        // pos2(1) = joint_positions(i*jpns+j+offset1,1);
                        // pos2(2) = joint_positions(i*jpns+j+offset1,2);
                        int curr_row = first_row+j+1;
                        pos2(0) = joint_positions(curr_row,0);
                        pos2(1) = joint_positions(curr_row,1);
                        pos2(2) = joint_positions(curr_row,2);

                        // if (k == 0){
                        //     if (i == 0){
                        //         std::cout << pos1(0) << ' ' << pos1(1) << ' ' << pos1(2) << std::endl;
                        //         std::cout << pos2(0) << ' ' << pos2(1) << ' ' << pos2(2) << std::endl;
                        //         std::cout << first_row << ' ' << curr_row << std::endl;
                        //         std::cout << std::endl;
                        //     }
                        // }
                        // if (k == 1){
                        //     if (i == 0){
                        //         std::cout << pos1(0) << ' ' << pos1(1) << ' ' << pos1(2) << std::endl;
                        //         std::cout << pos2(0) << ' ' << pos2(1) << ' ' << pos2(2) << std::endl;
                        //         std::cout << first_row << ' ' << curr_row << std::endl;
                        //         std::cout << std::endl;
                        //     }
                        // }
                        // if (k == 2){
                        //     if (i == 0){
                        //         std::cout << pos1(0) << ' ' << pos1(1) << ' ' << pos1(2) << std::endl;
                        //         std::cout << pos2(0) << ' ' << pos2(1) << ' ' << pos2(2) << std::endl;
                        //         std::cout << first_row << ' ' << curr_row << std::endl;
                        //         std::cout << std::endl;
                        //     }
                        // }
                        
                        // form the link zonotope
                        link_sliced_center[i * NUM_JOINTS + j] = 0.5 * (pos1 + pos2);
                        link_independent_generators[i * NUM_JOINTS + j] = 0.5 * (pos1 - pos2);
                        // TODO : ask Bohao why these indices are different than below loop
                        
                        // update previous position
                        pos1 = pos2;
                    }
                }

                /*
                Section II: Buffer obstacles and initialize collision checking hyperplanes
                */
                try {
                    O_ptr2->initializeHyperPlane(link_independent_generators);
                }
                catch (int errorCode) {
                    WARNING_PRINT("        CUDA & C++: Error initializing collision checking hyperplanes! Check previous error message!");
                    return new_adj_matrix; // -1; // 
                }

                /*
                Section III:
                    Collision checking
                */

                try {
                    O_ptr2->linkFRSConstraints(link_sliced_center, link_c);
                }
                catch (int errorCode) {std::
                    WARNING_PRINT("        CUDA & C++: Error peforming collision checking! Check previous error message!");
                    return new_adj_matrix; // -1; // 
                }

                /*
                Section IV:
                    Prepare output
                */
                // for (int i = 0; i < NUM_NODES_AT_ONE_TIME * num_obstacles; i++) {
                //     for (int j = 0; j < NUM_JOINTS; j++) {
                //         outputstream2 << link_c[j * NUM_NODES_AT_ONE_TIME * num_obstacles + i] << ' ';
                //     }
                //     outputstream2 << '\n';
                // }
                for (int i = 0; i < NUM_NODES_AT_ONE_TIME; i++) {
                    bool node_feasibility = true;
                    for (int j = 0; j < NUM_JOINTS; j++) {
                        for (int h = 0; h < num_obstacles; h++) {
                            if (link_c[(j * NUM_NODES_AT_ONE_TIME + i) * num_obstacles + h] > COLLISION_THRESHOLD) {
                                node_feasibility = false;
                                break;
                            }
                        }
                    }
                    // outputstream1 << node_feasibility << endl;
                    node_feasibilities[k * NUM_NODES_AT_ONE_TIME + i] = node_feasibility;
                }
            }

            auto stop1 = std::chrono::high_resolution_clock::now();
            auto duration1 = std::chrono::duration_cast<std::chrono::milliseconds>(stop1 - start1);
            cout << "Time taken by peforming collision checking: " << duration1.count() << " milliseconds" << endl;

            // building adjacency matrix
            // std::ifstream inputstream3(inputfilename3);
            // std::ofstream outputstream3(outputfilename3);

            // for checking if specific nodes are feasible
            // if (!node_feasibilities[0]) {
            //     // checking 1st node for feasibility as start in hardware
            //     std::string output1 = "Start Node is infeasible";
            //     std::cout << std::endl << output1 << std::endl;
            // }
            // if (!node_feasibilities[1]) {
            //     // checking 2nd node for feasibility as goal in hardware
            //     std::string output2 = "Goal Node is infeasible";
            //     std::cout << std::endl << output2 << std::endl;
            // }
            // if (!node_feasibilities[13]) {
            //     // checking 20th node for feasibility as goal in hardware
            //     std::string output3 = "New Goal Node is infeasible";
            //     std::cout << std::endl << output3 << std::endl;
            // }
            
            auto start2 = std::chrono::high_resolution_clock::now();

            // build the new adjacency matrix
            for (int i = 0; i < NUM_EDGES; i++) {
                int node_a = old_adj_matrix(i,0);
                int node_b = old_adj_matrix(i,1);
                double edge_distance = old_adj_matrix(i,2);
                // inputstream3 >> node_a >> node_b >> edge_distance; // TODO replace this when passing in old adjacency matrix

                if (node_feasibilities[node_a] && node_feasibilities[node_b] && edge_distance < EDGE_THRESHOLD) {
                    // new_adj_nodes[i,0] = node_a;
                    // new_adj_nodes[i,1] = node_b;
                    // new_edge_distance[i,0] = edge_distance;

                    // std::cout << node_a << ' ' << node_b << ' ' << edge_distance << std::endl;
                    // std::cout << static_cast<float>(node_a) << ' ' << static_cast<float>(node_b) << ' ' << edge_distance << std::endl << std::endl;

                    // new_adj_matrix(i,0) = static_cast<float>(node_a);
                    // new_adj_matrix(i,1) = static_cast<float>(node_b); // trying to cast
                    // new_adj_matrix(i,2) = edge_distance;

                    new_adj_matrix_ptr[3*i] = float(node_a);
                    new_adj_matrix_ptr[3*i+1] = float(node_b); // trying to cast // _ptr
                    new_adj_matrix_ptr[3*i+2] = edge_distance;

                    // std::string output2 = "Feasible Node";
                    // std::cout << output2 << std::endl;

                    // outputstream3 << node_a << ' ' << node_b << ' ' << edge_distance << '\n';
                }
            }
            // could potentially parallelize the building of the adjacency matrix if not outputting to file?

            auto stop2 = std::chrono::high_resolution_clock::now();
            auto duration2 = std::chrono::duration_cast<std::chrono::milliseconds>(stop2 - start2);
            cout << "Time taken by building new adjacency matrix: " << duration2.count() << " milliseconds" << endl;

            // inputstream2.close();
            // inputstream3.close();
            // outputstream1.close();
            // outputstream2.close();
            // outputstream3.close();
            delete[] link_c;
            delete[] node_feasibilities;

            return new_adj_matrix; // 0; // 

        }

        void free()
        {
            O_ptr.reset();
        }

        void set_datadir(const std::string &dir)
        {
            data_dir = dir;
        }

        std::string get_datadir()
        {
            return data_dir;
        }

};

PYBIND11_MODULE(armour_main_pybind, m) {
    py::class_<pzsparse>(m, "pzsparse")
        .def(py::init<py::array_t<double> &, const std::string &>())
        .def("getNumObstacles", &pzsparse::getNumObstacles)
        .def("optimize", &pzsparse::optimize)
        .def("getDesTraj", &pzsparse::getDesTraj)
        .def("setDataDir", &pzsparse::set_datadir)
        .def("getDataDir", &pzsparse::get_datadir)
        .def("graphCollisionCheck", &pzsparse::graphCollisionCheck, py::return_value_policy::copy); // reference_internal
        // .def("free", &pzsparse::free);
}