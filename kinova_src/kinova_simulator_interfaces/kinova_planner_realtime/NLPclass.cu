#include "hip/hip_runtime.h"
#ifndef NLP_CLASS_CU
#define NLP_CLASS_CU

#include "NLPclass.h"

// constructor
armtd_NLP::armtd_NLP()
{
    checkJointsPosition = new TYPE[NUM_TIME_STEPS * NUM_FACTORS * 3];
    dk_checkJointsPosition = new TYPE[NUM_TIME_STEPS * NUM_FACTORS * 3 * NUM_FACTORS];
}


// destructor
armtd_NLP::~armtd_NLP()
{
    delete[] checkJointsPosition;
    delete[] dk_checkJointsPosition;
    delete[] g_copy;
}


bool armtd_NLP::set_parameters(
    TYPE* q_des_input,
    TYPE t_plan_input,
    BezierCurve* desired_trajectory_input,
    PZsparse* joint_position_input,
    PZsparse* control_input_input,
    TYPE* v_norm_input,
    Obstacles* obstacles_input,
    vecPZsparse* f_c_input,
    vecPZsparse* n_c_input,
    const Number* u_s,
    const Number* surf_rad
 ) 
 {
    q_des = q_des_input;
    t_plan = t_plan_input;
    desired_trajectory = desired_trajectory_input;
    joint_position = joint_position_input;
    control_input = control_input_input;
    v_norm = v_norm_input;
    obstacles = obstacles_input;
    f_c = f_c_input;
    n_c = n_c_input;

    constraint_number = NUM_FACTORS * NUM_TIME_STEPS +
                        (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles + 
                        NUM_FACTORS * 4 + 
                        NUM_TIME_STEPS * 3;
    // 1. torque constraints
    // 2. collision checking constraints (ignoring base link and no self-collision)
    // 3. joint limit constraints, pos/vel lower and upper constraints
    // 4. force constraints

    g_copy = new Number[constraint_number];

    return true;
}


bool armtd_NLP::get_nlp_info(
   Index&          n,
   Index&          m,
   Index&          nnz_jac_g,
   Index&          nnz_h_lag,
   IndexStyleEnum& index_style
)
{
    // The problem described NUM_FACTORS variables, x[NUM_FACTORS] through x[NUM_FACTORS] for each joint
    n = NUM_FACTORS;

    // number of inequality constraint
    m = constraint_number;

    nnz_jac_g = m * n;

    // use the C style indexing (0-based)
    index_style = TNLP::C_STYLE;

    return true;
}
// [TNLP_get_nlp_info]

// [TNLP_get_bounds_info]
// returns the variable bounds
bool armtd_NLP::get_bounds_info(
   Index   n,
   Number* x_l,
   Number* x_u,
   Index   m,
   Number* g_l,
   Number* g_u
)
{
    // here, the n and m we gave IPOPT in get_nlp_info are passed back to us.
    // If desired, we could assert to make sure they are what we think they are.
    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in get_bounds_info!");
    }
    if(m != constraint_number){
        WARNING_PRINT("*** Error wrong value of m in get_bounds_info!");
    }

    // lower bounds
    for( Index i = 0; i < n; i++ ) {
        x_l[i] = -1.0;
    }

    // upper bounds  
    for( Index i = 0; i < n; i++ ) {
        x_u[i] = 1.0;
    }

    // control input constraints
    for( Index i = 0; i < NUM_TIME_STEPS; i++ ) {
        for( Index j = 0; j < NUM_FACTORS; j++ ) {
            g_l[i * NUM_FACTORS + j] = -torque_limits[j] + v_norm[i * NUM_FACTORS + j];
            g_u[i * NUM_FACTORS + j] = torque_limits[j] - v_norm[i * NUM_FACTORS + j];
        }
    }    
    Index offset = NUM_FACTORS * NUM_TIME_STEPS;

    // force constraints on contact joint between tray and cup
    // need to add the radius of the constraints here (like how is done with the torque constraints, v_norm passes out the radius I think)
    // double check signs and how the torque constraints are buffered using the radius
    // for v_norm in armour_main.cpp : getRadius(u_nom[t_ind * NUM_FACTORS + i].independent)

    //     separation constraint
    // upper bound should be zero and lower bound should be -inf
    for( Index i = offset; i < offset + NUM_TIME_STEPS; i++){
        g_l[i] = -1e19;
        g_u[i] = 0;
    }
    offset += NUM_TIME_STEPS;

    //     slipping constraint
    // upper bound should be zero and lower bound should be -inf for the reformulated constraint (not the normal friction law?)
    for( Index i = offset; i < offset + NUM_TIME_STEPS; i++){
        g_l[i] = -1e19;
        g_u[i] = 0;
    }
    offset += NUM_TIME_STEPS;

    //     tipping constraint
    // upper bound should be zero and lower bound should be -inf for the reformulated constraint
    for( Index i = offset; i < offset + NUM_TIME_STEPS; i++){
        g_l[i] = -1e19;
        g_u[i] = 0;
    }
    offset += NUM_TIME_STEPS;

    // collision avoidance constraints
    for( Index i = offset; i < offset + (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles; i++ ) {
        g_l[i] = -1e19;
        g_u[i] = 0;
    }
    offset += (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles;

    // state limit constraints
    //     minimum joint position
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        g_l[i] = state_limits_lb[i - offset] + qe;
        g_u[i] = state_limits_ub[i - offset] - qe;
    }
    offset += NUM_FACTORS;

    //     maximum joint position
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        g_l[i] = state_limits_lb[i - offset] + qe;
        g_u[i] = state_limits_ub[i - offset] - qe;
    }
    offset += NUM_FACTORS;

    //     minimum joint velocity
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        g_l[i] = -speed_limits[i - offset] + qde;
        g_u[i] = speed_limits[i - offset] - qde;
    }
    offset += NUM_FACTORS;

    //     maximum joint velocity
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        g_l[i] = -speed_limits[i - offset] + qde;
        g_u[i] = speed_limits[i - offset] - qde;
    }
    offset += NUM_FACTORS;

    return true;
}
// [TNLP_get_bounds_info]

// [TNLP_get_starting_point]
// returns the initial point for the problem
bool armtd_NLP::get_starting_point(
    Index   n,
    bool    init_x,
    Number* x,
    bool    init_z,
    Number* z_L,
    Number* z_U,
    Index   m,
    bool    init_lambda,
    Number* lambda
)
{
    // Here, we assume we only have starting values for x, if you code
    // your own NLP, you can provide starting values for the dual variables
    // if you wish
    if(init_x == false || init_z == true || init_lambda == true){
        WARNING_PRINT("*** Error wrong value of init in get_starting_point!");
    }

    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in get_starting_point!");
    }

    for( Index i = 0; i < n; i++ ) {
        // initialize to zero
        // x[i] = 0.0;

        // try to avoid local minimum
        x[i] = min(max((q_des[i] - desired_trajectory->q0[i]) / k_range[i], -0.5), 0.5);
    }

    return true;
}
// [TNLP_get_starting_point]

// [TNLP_eval_f]
// returns the value of the objective function
bool armtd_NLP::eval_f(
   Index         n,
   const Number* x,
   bool          new_x,
   Number&       obj_value
)
{
    if(n != NUM_FACTORS){
       WARNING_PRINT("*** Error wrong value of n in eval_f!");
    }

    // obj_value = sum((q_plan - q_des).^2);
    obj_value = 0; 
    for(Index i = 0; i < n; i++){
        TYPE q_plan = q_des_func(desired_trajectory->q0[i], desired_trajectory->qd0[i], desired_trajectory->qdd0[i], k_range[i] * x[i], t_plan);
        obj_value += pow(q_plan - q_des[i], 2);
    }

    obj_value *= 100.0;

    return true;
}
// [TNLP_eval_f]

// [TNLP_eval_grad_f]
// return the gradient of the objective function grad_{x} f(x)
bool armtd_NLP::eval_grad_f(
   Index         n,
   const Number* x,
   bool          new_x,
   Number*       grad_f
)
{
    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in eval_grad_f!");
    }

    for(Index i = 0; i < n; i++){
        TYPE q_plan = q_des_func(desired_trajectory->q0[i], desired_trajectory->qd0[i], desired_trajectory->qdd0[i], k_range[i] * x[i], t_plan);
        TYPE dk_q_plan = pow(t_plan,3) * (6 * pow(t_plan,2) - 15 * t_plan + 10);
        grad_f[i] = (2 * (q_plan - q_des[i]) * dk_q_plan * k_range[i]) * 100.0;
    }

    return true;
}
// [TNLP_eval_grad_f]

// [TNLP_eval_g]
// return the value of the constraints: g(x)
bool armtd_NLP::eval_g(
   Index         n,
   const Number* x,
   bool          new_x,
   Index         m,
   Number*       g
)
{
    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in eval_g!");
    }
    if(m != constraint_number){
        WARNING_PRINT("*** Error wrong value of m in eval_g!");
    }

    Index i;
    #pragma omp parallel for private(i) schedule(static, (NUM_TIME_STEPS * NUM_FACTORS + NUM_TIME_STEPS) / NUM_THREADS)
    for(i = 0; i < (NUM_TIME_STEPS * NUM_FACTORS + NUM_TIME_STEPS); i++) {  // took out the times 3 here since I am calculating all three contact constraints at the same time, but they still fill up all the same space
        
        if(i < (NUM_TIME_STEPS * NUM_FACTORS)) {
            // Part 1. slice the control input PZ to get the center of the input bound, 
            //         while the radius of the input bound has already been incorporated in ipopt constraint bounds
            g[i] = getCenter(control_input[i].slice(x));

            // Part 2. slice the forward kinematics PZ to get the center of the joint position bound, 
            //         while the radius of the joint position bound has already been considered in Obstacles class
            checkJointsPosition[i * 3    ] = getCenter(joint_position[i * 3    ].slice(x));
            checkJointsPosition[i * 3 + 1] = getCenter(joint_position[i * 3 + 1].slice(x));
            checkJointsPosition[i * 3 + 2] = getCenter(joint_position[i * 3 + 2].slice(x));
        } 
        else {
            // Part 3. force constraints on contact joint between tray and object
            // note: could change the next couple of lines to not slice multiple times

            //     force
            // get centers and their squares
            Number f_c_x_center = getCenter(f_c[i].elt[0].slice(x));
            Number f_c_x_center_2 = f_c_x_center * f_c_x_center;
            Number f_c_y_center = getCenter(f_c[i].elt[1].slice(x));
            Number f_c_y_center_2 = f_c_y_center * f_c_y_center;
            Number f_c_z_center = getCenter(f_c[i].elt[2].slice(x));
            Number f_c_z_center_2 = f_c_z_center * f_c_z_center;
            // get radii of independent generators and their squares
            TYPE f_c_x_radius = getRadius(f_c[i].elt[0].slice(x).independent)
            TYPE f_c_x_radius_2 = f_c_x_radius * f_c_x_radius;
            TYPE f_c_y_radius = getRadius(f_c[i].elt[1].slice(x).independent)
            TYPE f_c_y_radius_2 = f_c_y_radius * f_c_y_radius;
            TYPE f_c_z_radius = getRadius(f_c[i].elt[2].slice(x).independent)
            TYPE f_c_z_radius_2 = f_c_z_radius * f_c_z_radius;

            //     moment
            // get centers and their squares
            Number n_c_x_center = getCenter(n_c[i].elt[0].slice(x));
            Number n_c_x_center_2 = n_c_x_center * n_c_x_center;
            Number n_c_y_center = getCenter(n_c[i].elt[1].slice(x));
            Number n_c_y_center_2 = n_c_y_center * n_c_y_center;
            Number n_c_z_center = getCenter(n_c[i].elt[2].slice(x));
            Number n_c_z_center_2 = n_c_z_center * n_c_z_center;
            // get radii of independent generators and their squares
            TYPE n_c_x_radius = getRadius(n_c[i].elt[0].slice(x).independent)
            TYPE n_c_x_radius_2 = n_c_x_radius * n_c_x_radius;
            TYPE n_c_y_radius = getRadius(n_c[i].elt[1].slice(x).independent)
            TYPE n_c_y_radius_2 = n_c_y_radius * n_c_y_radius;
            TYPE n_c_z_radius = getRadius(n_c[i].elt[2].slice(x).independent)
            TYPE n_c_z_radius_2 = n_c_z_radius * n_c_z_radius;

            //     separation constraint: -inf < -1*f_c_z < 0
            // storing separation constraint value, not sure what index to use here?
            // question: not sure what to do with the radius here?
            // g[i] = -1*getCenter(f_c_z);
            g[i] = -1*f_c_z_center;

            //     slipping constraint: -inf < f_c_x*f_c_x + f_c_y*f_c_y - u_s^2*f_c_z*f_c_z < 0
            // need to write this constraint differently than matlab implementation as we need to 
            // slice before doing multiplications of PZs in order to avoid memory problems.
            // getCenter, getRadius (for v_norm in armour_main.cpp : getRadius(u_nom[t_ind * NUM_FACTORS + i].independent))

            // if ( getCenter(f_c[i].elt[0].slice(x)) >= 0) && (getCenter(f_c[i].elt[1].slice(x)) >= 0) && (getCenter(f_c[i].elt[2].slice(x)) >= 0 ){
            //     g[i+NUM_TIME_STEPS] = (getCenter(f_c[i].elt[0].slice(x)))_2 + 2*getRadius(f_c[i].elt[0].slice(x).independent)*getCenter(f_c[i].elt[0].slice(x)) + (getRadius(f_c[i].elt[0].slice(x).independent))_2 + (getCenter(f_c[i].elt[1].slice(x)))_2 + 2*getRadius(f_c[i].elt[1].slice(x).independent)*getCenter(f_c[i].elt[1].slice(x)) + (getRadius(f_c[i].elt[1].slice(x).independent))_2 - u_s_2*((getCenter(f_c[i].elt[2].slice(x)))_2 - 2*getRadius(f_c[i].elt[2].slice(x).independent)*getCenter(f_c[i].elt[2].slice(x)) - (getRadius(f_c[i].elt[2].slice(x).independent))_2);
            
            // check the signs of the centers of the force zonotopes
            // condition 1: all positive
            if ( (f_c_x_center >= 0) && (f_c_y_center >= 0) && (f_c_z_center >= 0) ){
                // Note: double check that the center/radius is a number that can be squared
                g[i+NUM_TIME_STEPS] = f_c_x_center_2 + 2*f_c_x_radius*f_c_x_center + f_c_x_radius_2 + f_c_y_center_2 + 2*f_c_y_radius*f_c_y_center + f_c_y_radius_2 - u_s*u_s * ( f_c_z_center_2 - 2*f_c_z_radius*f_c_z_center - f_c_z_radius_2); // checked signs
            }
            // condition 2: y negative
            else if ( (f_c_x_center >= 0) && (f_c_y_center <= 0) && (f_c_z_center >= 0) ) {
                g[i+NUM_TIME_STEPS] = f_c_x_center_2 + 2*f_c_x_radius*f_c_x_center + f_c_x_radius_2 + f_c_y_center_2 - 2*f_c_y_radius*f_c_y_center + f_c_y_radius_2 - u_s*u_s * ( f_c_z_center_2 - 2*f_c_z_radius*f_c_z_center - f_c_z_radius_2); // checked signs
            }
            // condition 3: z negative
            else if ( (f_c_x_center >= 0) && (f_c_y_center >= 0) && (f_c_z_center <= 0) ) {
                g[i+NUM_TIME_STEPS] = f_c_x_center_2 + 2*f_c_x_radius*f_c_x_center + f_c_x_radius_2 + f_c_y_center_2 + 2*f_c_y_radius*f_c_y_center + f_c_y_radius_2 - u_s*u_s * ( f_c_z_center_2 + 2*f_c_z_radius*f_c_z_center - f_c_z_radius_2); // checked signs
            }
            // condition 4: y and z negative
            else if ( (f_c_x_center >= 0) && (f_c_y_center <= 0) && (f_c_z_center <= 0) ) {
                g[i+NUM_TIME_STEPS] = f_c_x_center_2 + 2*f_c_x_radius*f_c_x_center + f_c_x_radius_2 + f_c_y_center_2 - 2*f_c_y_radius*f_c_y_center + f_c_y_radius_2 - u_s*u_s * ( f_c_z_center_2 + 2*f_c_z_radius*f_c_z_center - f_c_z_radius_2); // checked signs
            }
            // condition 5: x negative
            else if ( (f_c_x_center <= 0) && (f_c_y_center >= 0) && (f_c_z_center >= 0) ) {
                g[i+NUM_TIME_STEPS] = f_c_x_center_2 - 2*f_c_x_radius*f_c_x_center + f_c_x_radius_2 + f_c_y_center_2 + 2*f_c_y_radius*f_c_y_center + f_c_y_radius_2 - u_s*u_s * ( f_c_z_center_2 - 2*f_c_z_radius*f_c_z_center - f_c_z_radius_2); // checked signs
            }
            // condition 6: x and y negative
            else if ( (f_c_x_center <= 0) && (f_c_y_center <= 0) && (f_c_z_center >= 0) ) {
                g[i+NUM_TIME_STEPS] = f_c_x_center_2 - 2*f_c_x_radius*f_c_x_center + f_c_x_radius_2 + f_c_y_center_2 - 2*f_c_y_radius*f_c_y_center + f_c_y_radius_2 - u_s*u_s * ( f_c_z_center_2 - 2*f_c_z_radius*f_c_z_center - f_c_z_radius_2); // checked signs
            }
            // condition 7: x and z negative
            else if ( (f_c_x_center <= 0) && (f_c_y_center >= 0) && (f_c_z_center <= 0) ) {
                g[i+NUM_TIME_STEPS] = f_c_x_center_2 - 2*f_c_x_radius*f_c_x_center + f_c_x_radius_2 + f_c_y_center_2 + 2*f_c_y_radius*f_c_y_center + f_c_y_radius_2 - u_s*u_s * ( f_c_z_center_2 + 2*f_c_z_radius*f_c_z_center - f_c_z_radius_2); // checked signs
            }
            // condition 8: x and y and z negative
            else if ( (f_c_x_center <= 0) && (f_c_y_center <= 0) && (f_c_z_center <= 0) ) {
                g[i+NUM_TIME_STEPS] = f_c_x_center_2 - 2*f_c_x_radius*f_c_x_center + f_c_x_radius_2 + f_c_y_center_2 - 2*f_c_y_radius*f_c_y_center + f_c_y_radius_2 - u_s*u_s * ( f_c_z_center_2 + 2*f_c_z_radius*f_c_z_center - f_c_z_radius_2); // checked signs
            }

            //     tipping constraint: ZMP_top_x*ZMP_top_x + ZMP_top_y*ZMP_top_y - surf_rad*ZMP_bottom*ZMP_bottom < 0

            // compute the numerator of the ZMP point equation
            vecPZsparse ZMP_top = cross([0;0;1],n_c[i]);
            // extract the x, y and z components, slice by the parameters, then get the centers and radii of independent generators and their squares
            // x-component
            PZsparse ZMP_top_x = ZMP_top.elt[0].slice(x);
            ZMP_top_x_center = getCenter(ZMP_top_x);
            ZMP_top_x_center_2 = ZMP_top_x_center*ZMP_top_x_center;
            ZMP_top_x_radius = getRadius(ZMP_top_x.independent);
            ZMP_top_x_radius_2 = ZMP_top_x_radius*ZMP_top_x_radius;
            // y-component
            ZMP_top_y = ZMP_top.elt[1].slice(x);
            ZMP_top_y_center = getCenter(ZMP_top_y);
            ZMP_top_y_center_2 = ZMP_top_y_center*ZMP_top_y_center;
            ZMP_top_y_radius = getRadius(ZMP_top_y.independent);
            ZMP_top_y_radius_2 = ZMP_top_y_radius*ZMP_top_y_radius;
            // z-component
            ZMP_top_z = ZMP_top.elt[2].slice(x); // use for debugging, check this is always equal to zero
            
            // compute the denominator of the ZMP point equation
            // note that if the normal vector corresponds to the body frame z-axis, n=[0;0;1] and the dot product of that normal vector
            // with the moment results in the z-component of the moment.
            // question: is the moment about the contact point like in Matlab RNEA? need to verify this
            ZMP_bottom = n_c[i].elt[2].slice(x);
            // extract center and radius of independent generators and their squares
            ZMP_bottom_center = getCenter(ZMP_bottom);
            ZMP_bottom_center_2 = ZMP_bottom_center*ZMP_bottom_center;
            ZMP_bottom_radius = getRadius(ZMP_bottom.independent);
            ZMP_bottom_radius_2 = ZMP_bottom_radius*ZMP_bottom_radius;
            
            // check the signs of the centers of the force zonotopes in order to form the constraint
            // condition 1: all positive
            if ( (ZMP_top_x_center >= 0) && (ZMP_top_y_center >= 0) && (ZMP_bottom_center >= 0) ){
                // Note: double check that the center/radius is a number that can be squared
                g[i+2*NUM_TIME_STEPS] = ZMP_top_x_center_2 + 2*ZMP_top_x_radius*ZMP_top_x_center + ZMP_top_x_radius_2 + ZMP_top_y_center_2 + 2*ZMP_top_y_radius*ZMP_top_y_center + ZMP_top_y_radius_2 - surf_rad_2 * ( ZMP_bottom_center_2 - 2*ZMP_bottom_radius*ZMP_bottom_center - ZMP_bottom_radius_2); // checked signs
            }
            // condition 2: y negative
            else if ( (ZMP_top_x_center >= 0) && (ZMP_top_y_center <= 0) && (ZMP_bottom_center >= 0) ) {
                g[i+2*NUM_TIME_STEPS] = ZMP_top_x_center_2 + 2*ZMP_top_x_radius*ZMP_top_x_center + ZMP_top_x_radius_2 + ZMP_top_y_center_2 - 2*ZMP_top_y_radius*ZMP_top_y_center + ZMP_top_y_radius_2 - surf_rad_2 * ( ZMP_bottom_center_2 - 2*ZMP_bottom_radius*ZMP_bottom_center - ZMP_bottom_radius_2); // checked signs
            }
            // condition 3: z negative
            else if ( (ZMP_top_x_center >= 0) && (ZMP_top_y_center >= 0) && (ZMP_bottom_center <= 0) ) {
                g[i+2*NUM_TIME_STEPS] = ZMP_top_x_center_2 + 2*ZMP_top_x_radius*ZMP_top_x_center + ZMP_top_x_radius_2 + ZMP_top_y_center_2 + 2*ZMP_top_y_radius*ZMP_top_y_center + ZMP_top_y_radius_2 - surf_rad_2 * ( ZMP_bottom_center_2 + 2*ZMP_bottom_radius*ZMP_bottom_center - ZMP_bottom_radius_2); // checked signs
            }
            // condition 4: y and z negative
            else if ( (ZMP_top_x_center >= 0) && (ZMP_top_y_center <= 0) && (ZMP_bottom_center <= 0) ) {
                g[i+2*NUM_TIME_STEPS] = ZMP_top_x_center_2 + 2*ZMP_top_x_radius*ZMP_top_x_center + ZMP_top_x_radius_2 + ZMP_top_y_center_2 - 2*ZMP_top_y_radius*ZMP_top_y_center + ZMP_top_y_radius_2 - surf_rad_2 * ( ZMP_bottom_center_2 + 2*ZMP_bottom_radius*ZMP_bottom_center - ZMP_bottom_radius_2); // checked signs
            }
            // condition 5: x negative
            else if ( (ZMP_top_x_center <= 0) && (ZMP_top_y_center >= 0) && (ZMP_bottom_center >= 0) ) {
                g[i+2*NUM_TIME_STEPS] = ZMP_top_x_center_2 - 2*ZMP_top_x_radius*ZMP_top_x_center + ZMP_top_x_radius_2 + ZMP_top_y_center_2 + 2*ZMP_top_y_radius*ZMP_top_y_center + ZMP_top_y_radius_2 - surf_rad_2 * ( ZMP_bottom_center_2 - 2*ZMP_bottom_radius*ZMP_bottom_center - ZMP_bottom_radius_2); // checked signs
            }
            // condition 6: x and y negative
            else if ( (ZMP_top_x_center <= 0) && (ZMP_top_y_center <= 0) && (ZMP_bottom_center >= 0) ) {
                g[i+2*NUM_TIME_STEPS] = ZMP_top_x_center_2 - 2*ZMP_top_x_radius*ZMP_top_x_center + ZMP_top_x_radius_2 + ZMP_top_y_center_2 - 2*ZMP_top_y_radius*ZMP_top_y_center + ZMP_top_y_radius_2 - surf_rad_2 * ( ZMP_bottom_center_2 - 2*ZMP_bottom_radius*ZMP_bottom_center - ZMP_bottom_radius_2); // checked signs
            }
            // condition 7: x and z negative
            else if ( (ZMP_top_x_center <= 0) && (ZMP_top_y_center >= 0) && (ZMP_bottom_center <= 0) ) {
                g[i+2*NUM_TIME_STEPS] = ZMP_top_x_center_2 - 2*ZMP_top_x_radius*ZMP_top_x_center + ZMP_top_x_radius_2 + ZMP_top_y_center_2 + 2*ZMP_top_y_radius*ZMP_top_y_center + ZMP_top_y_radius_2 - surf_rad_2 * ( ZMP_bottom_center_2 + 2*ZMP_bottom_radius*ZMP_bottom_center - ZMP_bottom_radius_2); // checked signs
            }
            // condition 8: x and y and z negative
            else if ( (ZMP_top_x_center <= 0) && (ZMP_top_y_center <= 0) && (ZMP_bottom_center <= 0) ) {
                g[i+2*NUM_TIME_STEPS] = ZMP_top_x_center_2 - 2*ZMP_top_x_radius*ZMP_top_x_center + ZMP_top_x_radius_2 + ZMP_top_y_center_2 - 2*ZMP_top_y_radius*ZMP_top_y_center + ZMP_top_y_radius_2 - surf_rad_2 * ( ZMP_bottom_center_2 + 2*ZMP_bottom_radius*ZMP_bottom_center - ZMP_bottom_radius_2); // checked signs
            }

        }
        
    }

    // Part 4. check collision between joint position reachable set and obstacles (in gpu)
    obstacles->linkFRSConstraints(checkJointsPosition, nullptr, g + (NUM_TIME_STEPS * NUM_FACTORS + NUM_TIME_STEPS*3), nullptr);

    // Part 5. (position & velocity) state limit constraints
    desired_trajectory->returnJointPositionExtremum(g + (NUM_TIME_STEPS * NUM_FACTORS + NUM_TIME_STEPS*3) + (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles, x);
    desired_trajectory->returnJointVelocityExtremum(g + (NUM_TIME_STEPS * NUM_FACTORS + NUM_TIME_STEPS*3) + (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles + NUM_FACTORS * 2, x);

    return true;
}
// [TNLP_eval_g]


// [TNLP_eval_jac_g]
// return the structure or values of the Jacobian
bool armtd_NLP::eval_jac_g(
   Index         n,
   const Number* x,
   bool          new_x,
   Index         m,
   Index         nele_jac,
   Index*        iRow,
   Index*        jCol,
   Number*       values
)
{
    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in eval_g!");
    }
    if(m != constraint_number){
        WARNING_PRINT("*** Error wrong value of m in eval_g!");
    }
        
    if( values == NULL ) {
       // return the structure of the Jacobian
       // this particular Jacobian is dense
        for(Index i = 0; i < m; i++){
            for(Index j = 0; j < n; j++){
                iRow[i * n + j] = i;
                jCol[i * n + j] = j;
            }
        }
    }
    else {
        Index i;
        #pragma omp parallel for private(i) schedule(static, (NUM_TIME_STEPS * NUM_FACTORS + NUM_TIME_STEPS)/ NUM_THREADS)
        for(i = 0; i < (NUM_TIME_STEPS * NUM_FACTORS + NUM_TIME_STEPS); i++) { // not *3 for the contact constraints since they're being done at the same time.

            if (i < NUM_TIME_STEPS * NUM_FACTORS) {
            // Part 1. slice the control input PZ to get the center of the input bound, 
            //         while the radius of the input bound has already been incorporated in ipopt constraint bounds
            control_input[i].slice(values + i * NUM_FACTORS, x);

            // Part 2. slice the forward kinematics PZ to get the center of the joint position bound, 
            //         while the radius of the joint position bound has already been considered in Obstacles class
            checkJointsPosition[i * 3    ] = getCenter(joint_position[i * 3    ].slice(x));
            checkJointsPosition[i * 3 + 1] = getCenter(joint_position[i * 3 + 1].slice(x));
            checkJointsPosition[i * 3 + 2] = getCenter(joint_position[i * 3 + 2].slice(x));
            joint_position[i * 3    ].slice(dk_checkJointsPosition + (i * 3    ) * NUM_FACTORS, x);
            joint_position[i * 3 + 1].slice(dk_checkJointsPosition + (i * 3 + 1) * NUM_FACTORS, x);
            joint_position[i * 3 + 2].slice(dk_checkJointsPosition + (i * 3 + 2) * NUM_FACTORS, x);
            }
            else {
                // Part 3. Force constraints
                // how to properly index this part?

                // .slice(##, x) fills the pointer ## with the gradient, so for one instance it fills up by NUM_FACTORS


                // separation constraint
                // .slice(values + i*NUM_FACTORS, x)
                // should be able to use .slice(##, x) here
                f_c[i].elt[2].slice(values + i*NUM_FACTORS, x)

                // slipping constraint
                // .slice(values + (NUM_FACTORS*NUM_TIME_STEPS + NUM_TIME_STEPS)*NUM_FACTORS, x)
                values[(i+NUM_TIME_STEPS)*NUM_FACTORS] = ;
                // create function for return partial of a passed in index? function(g[i])
                for (int j=0; j<NUM_FACTORS; j++) {
                    for (int k=0; k<NUM_FACTORS; k++) {
                        if (j==k) {
                            if (degreeArray[j]==0) {

                            }
                            else {
                                
                            }
                        }
                        else {

                        }
                    }
                }

                //tipping constraint
                // .slice(, x)
                values[(i+2*NUM_TIME_STEPS)*NUM_FACTORS] = ; // need to fill in all the factors here
                
                
            }
            

        }

        // Part 4. check collision between joint position reachable set and obstacles (in gpu)
        obstacles->linkFRSConstraints(checkJointsPosition, dk_checkJointsPosition, nullptr, values + NUM_TIME_STEPS * NUM_FACTORS * NUM_FACTORS); // needs to be updated

        // Part 5. (position & velocity) state limit constraints
        desired_trajectory->returnJointPositionExtremumGradient(values + (NUM_TIME_STEPS * NUM_FACTORS + (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles) * NUM_FACTORS, x); // needs to be updated
        desired_trajectory->returnJointVelocityExtremumGradient(values + (NUM_TIME_STEPS * NUM_FACTORS + (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles + NUM_FACTORS * 2) * NUM_FACTORS, x); // needs to be updated
    }

    return true;
}
// [TNLP_eval_jac_g]


// void Number::contactConstraintPartial(TYPE* gradient, const Number contactEquation) {
//     // inputs: 
//     //    pointer to storage location?
//     //    constraint to take derivative of? (slip, tip? then have if statement for each)
//     //    pointer to f_c and n_c?

//     if ( contactEquation == 1 ) {
//         // take gradient of slip constraint equation wrt trajectory parameters
        
//     }
//     else if ( contactEquation == 2 ) {

//     }


// }


// [TNLP_eval_h]
//return the structure or values of the Hessian
bool armtd_NLP::eval_h(
   Index         n,
   const Number* x,
   bool          new_x,
   Number        obj_factor,
   Index         m,
   const Number* lambda,
   bool          new_lambda,
   Index         nele_hess,
   Index*        iRow,
   Index*        jCol,
   Number*       values
)
{
    return false;
}
// [TNLP_eval_h]


// [TNLP_finalize_solution]
void armtd_NLP::finalize_solution(
    SolverReturn               status,
    Index                      n,
    const Number*              x,
    const Number*              z_L,
    const Number*              z_U,
    Index                      m,
    const Number*              g,
    const Number*              lambda,
    Number                     obj_value,
    const IpoptData*           ip_data,
    IpoptCalculatedQuantities* ip_cq
)
{
    // here is where we would store the solution to variables, or write to a file, etc
    // so we could use the solution.

    // store the solution
    for( Index i = 0; i < n; i++ ) {
        solution[i] = (TYPE)x[i];
    }

    // check constraint violation manually for Maximum_CpuTime_Exceeded case
    memcpy(g_copy, g, m * sizeof(Number));

    feasible = true;

    // control input constraints
    for( Index i = 0; i < NUM_TIME_STEPS; i++ ) {
        for( Index j = 0; j < NUM_FACTORS; j++ ) {
            if (g_copy[i * NUM_FACTORS + j] < -torque_limits[j] + v_norm[i * NUM_FACTORS + j] - TORQUE_INPUT_CONSTRAINT_VIOLATION_THRESHOLD || 
                g_copy[i * NUM_FACTORS + j] > torque_limits[j] - v_norm[i * NUM_FACTORS + j] + TORQUE_INPUT_CONSTRAINT_VIOLATION_THRESHOLD) {
                feasible = false;
                cout << "        CUDA & C++: Ipopt: Control torque of joint " << j << " at time interval " << i << " exceeds limit!\n";
                cout << "                        value: " << g_copy[i * NUM_FACTORS + j] << "\n";
                cout << "                        range: [ " << -torque_limits[j] + v_norm[i * NUM_FACTORS + j] << ", "
                                                            << torque_limits[j] - v_norm[i * NUM_FACTORS + j] << " ]\n";
                return;
            }
        }
    }    

    // NOTE: need to add force constraints here and adjust the indices offsets after this
    Index offset = NUM_FACTORS * NUM_TIME_STEPS;
    for( Index i = offset; i < NUM_TIME_STEPS; i++) {
        // separation constraint
        if(g_copy[i] > SEPARATION_CONSTRAINT_VIOLATION_THRESHOLD) {
            feasible = false;
            double t_violation = i - offset;
            cout << "        CUDA & C++: Ipopt: Separation constraint violated at time interval: " << t_violation << " \n";
        }
        // slipping constraint
        if(g_copy[i+NUM_TIME_STEPS] > SLIPPING_CONSTRAINT_VIOLATION_THRESHOLD){
            feasible = false;
            double t_violation = i - offset;
            cout << "        CUDA & C++: Ipopt: Slipping constraint violated at time interval: " << t_violation << " \n";
        }
        // tipping constraint
        if(g_copy[i+2*NUM_TIME_STEPS] > TIPPING_CONSTRAINT_VIOLATION_THRESHOLD){
            feasible = false;
            double t_violation = i - offset;
            cout << "        CUDA & C++: Ipopt: Tipping constraint violated at time interval: " << t_violation << " \n";
        }
    }
    offset +=  NUM_TIME_STEPS*3;

    // collision avoidance constraints
    // Index offset = NUM_FACTORS * NUM_TIME_STEPS + NUM_TIME_STEPS*3;
    for( Index i = 0; i < NUM_FACTORS - 1; i++ ) {
        for( Index j = 0; j < NUM_TIME_STEPS; j++ ) {
            for( Index h = 0; h < obstacles->num_obstacles; h++ ) {
                if (g_copy[(i * NUM_TIME_STEPS + j) * obstacles->num_obstacles + h + offset] > COLLISION_AVOIDANCE_CONSTRAINT_VIOLATION_THRESHOLD) {
                    feasible = false;
                    cout << "        CUDA & C++: Ipopt: Collision between link " << i + 1 << " and obstacle " << h << " at time interval " << j << "!\n";
                    cout << "                        value: " << g_copy[(i * NUM_TIME_STEPS + j) * obstacles->num_obstacles + h + offset] << "\n";
                    return;
                }
            }
        }
    }
    offset += (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles;

    

    // state limit constraints
    //     minimum joint position
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        if (g_copy[i] < state_limits_lb[i - offset] + qe || g_copy[i] > state_limits_ub[i - offset] - qe) {
            feasible = false;
            cout << "        CUDA & C++: Ipopt: joint " << i - offset << " exceeds position limit when it reaches minimum!\n";
            cout << "                        value: " << g_copy[i] << "\n";
            cout << "                        range: [ " << state_limits_lb[i - offset] + qe << ", "
                                                        << state_limits_ub[i - offset] - qe << " ]\n";
            return;
        }
    }
    offset += NUM_FACTORS;

    //     maximum joint position
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        if (g_copy[i] < state_limits_lb[i - offset] + qe || g_copy[i] > state_limits_ub[i - offset] - qe) {
            feasible = false;
            cout << "        CUDA & C++: Ipopt: joint " << i - offset << " exceeds position limit when it reaches maximum!\n";
            cout << "                        value: " << g_copy[i] << "\n";
            cout << "                        range: [ " << state_limits_lb[i - offset] + qe << ", "
                                                        << state_limits_ub[i - offset] - qe << " ]\n";
            return;
        }
    }
    offset += NUM_FACTORS;

    //     minimum joint velocity
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        if (g_copy[i] < -speed_limits[i - offset] + qde || g_copy[i] > speed_limits[i - offset] - qde) {
            feasible = false;
            cout << "        CUDA & C++: Ipopt: joint " << i - offset << " exceeds velocity limit when it reaches minimum!\n";
            cout << "                        value: " << g_copy[i] << "\n";
            cout << "                        range: [ " << -speed_limits[i - offset] + qde << ", "
                                                        << speed_limits[i - offset] - qde << " ]\n";
            return;
        }
    }
    offset += NUM_FACTORS;

    //     maximum joint velocity
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        if (g_copy[i] < -speed_limits[i - offset] + qde || g_copy[i] > speed_limits[i - offset] - qde) {
            feasible = false;
            cout << "        CUDA & C++: Ipopt: joint " << i - offset << " exceeds velocity limit when it reaches maximum!\n";
            cout << "                        value: " << g_copy[i] << "\n";
            cout << "                        range: [ " << -speed_limits[i - offset] + qde << ", "
                                                        << speed_limits[i - offset] - qde << " ]\n";
            return;
        }
    }
}
// [TNLP_finalize_solution]


#endif
