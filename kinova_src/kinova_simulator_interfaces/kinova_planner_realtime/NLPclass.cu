#include "hip/hip_runtime.h"
#ifndef NLP_CLASS_CU
#define NLP_CLASS_CU

#include "NLPclass.h"

// constructor
armtd_NLP::armtd_NLP()
{
    checkJointsPosition = new TYPE[NUM_TIME_STEPS * NUM_FACTORS * 3];
    dk_checkJointsPosition = new TYPE[NUM_TIME_STEPS * NUM_FACTORS * 3 * NUM_FACTORS];
}


// destructor
armtd_NLP::~armtd_NLP()
{
    delete[] checkJointsPosition;
    delete[] dk_checkJointsPosition;
    delete[] g_copy;
}


bool armtd_NLP::set_parameters(
    TYPE* q_des_input,
    TYPE t_plan_input,
    BezierCurve* desired_trajectory_input,
    PZsparse* joint_position_input,
    PZsparse* control_input_input,
    TYPE* v_norm_input,
    Obstacles* obstacles_input,
    vecPZsparse* f_c_input,
    vecPZsparse* n_c_input
 ) 
 {
    q_des = q_des_input;
    t_plan = t_plan_input;
    desired_trajectory = desired_trajectory_input;
    joint_position = joint_position_input;
    control_input = control_input_input;
    v_norm = v_norm_input;
    obstacles = obstacles_input;
    f_c = f_c_input;
    n_c = n_c_input;

    constraint_number = NUM_FACTORS * NUM_TIME_STEPS +
                        (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles + 
                        NUM_FACTORS * 4 + 
                        NUM_TIME_STEPS * 3;
    // 1. torque constraints
    // 2. collision checking constraints (ignoring base link and no self-collision)
    // 3. joint limit constraints, pos/vel lower and upper constraints
    // 4. force constraints

    g_copy = new Number[constraint_number];

    return true;
}


bool armtd_NLP::get_nlp_info(
   Index&          n,
   Index&          m,
   Index&          nnz_jac_g,
   Index&          nnz_h_lag,
   IndexStyleEnum& index_style
)
{
    // The problem described NUM_FACTORS variables, x[NUM_FACTORS] through x[NUM_FACTORS] for each joint
    n = NUM_FACTORS;

    // number of inequality constraint
    m = constraint_number;

    nnz_jac_g = m * n;

    // use the C style indexing (0-based)
    index_style = TNLP::C_STYLE;

    return true;
}
// [TNLP_get_nlp_info]

// [TNLP_get_bounds_info]
// returns the variable bounds
bool armtd_NLP::get_bounds_info(
   Index   n,
   Number* x_l,
   Number* x_u,
   Index   m,
   Number* g_l,
   Number* g_u
)
{
    // here, the n and m we gave IPOPT in get_nlp_info are passed back to us.
    // If desired, we could assert to make sure they are what we think they are.
    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in get_bounds_info!");
    }
    if(m != constraint_number){
        WARNING_PRINT("*** Error wrong value of m in get_bounds_info!");
    }

    // lower bounds
    for( Index i = 0; i < n; i++ ) {
        x_l[i] = -1.0;
    }

    // upper bounds  
    for( Index i = 0; i < n; i++ ) {
        x_u[i] = 1.0;
    }

    // control input constraints
    for( Index i = 0; i < NUM_TIME_STEPS; i++ ) {
        for( Index j = 0; j < NUM_FACTORS; j++ ) {
            g_l[i * NUM_FACTORS + j] = -torque_limits[j] + v_norm[i * NUM_FACTORS + j];
            g_u[i * NUM_FACTORS + j] = torque_limits[j] - v_norm[i * NUM_FACTORS + j];
        }
    }    
    Index offset = NUM_FACTORS * NUM_TIME_STEPS;

    // collision avoidance constraints
    for( Index i = offset; i < offset + (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles; i++ ) {
        g_l[i] = -1e19;
        g_u[i] = 0;
    }
    offset += (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles;

    // state limit constraints
    //     minimum joint position
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        g_l[i] = state_limits_lb[i - offset] + qe;
        g_u[i] = state_limits_ub[i - offset] - qe;
    }
    offset += NUM_FACTORS;

    //     maximum joint position
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        g_l[i] = state_limits_lb[i - offset] + qe;
        g_u[i] = state_limits_ub[i - offset] - qe;
    }
    offset += NUM_FACTORS;

    //     minimum joint velocity
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        g_l[i] = -speed_limits[i - offset] + qde;
        g_u[i] = speed_limits[i - offset] - qde;
    }
    offset += NUM_FACTORS;

    //     maximum joint velocity
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        g_l[i] = -speed_limits[i - offset] + qde;
        g_u[i] = speed_limits[i - offset] - qde;
    }
    offset += NUM_FACTORS;



    // force constraints on contact joint between tray and cup
    // need to add the radius of the constraints here (like how is done with the torque constraints, v_norm passes out the radius I think)
    // double check signs and how the torque constraints are buffered using the radius
    // for v_norm in armour_main.cpp : getRadius(u_nom[t_ind * NUM_FACTORS + i].independent)


    //     separation constraint
    // upper bound should be zero and lower bound should be -inf
    for( Index i = offset; i < offset + NUM_TIME_STEPS; i++){
        g_l[i] = -1e19;
        g_u[i] = 0;
    }
    offset += NUM_TIME_STEPS;

    //     slipping constraint
    // upper bound should be zero and lower bound should be -inf for the reformulated constraint (not the normal friction law?)
    for( Index i = offset; i < offset + NUM_TIME_STEPS; i++){
        g_l[i] = -1e19;
        g_u[i] = 0;
    }
    offset += NUM_TIME_STEPS;

    //     tipping constraint
    // upper bound should be zero and lower bound should be -inf for the reformulated constraint
    for( Index i = offset; i < offset + NUM_TIME_STEPS; i++){
        g_l[i] = -1e19;
        g_u[i] = 0;
    }
    offset += NUM_TIME_STEPS;

    return true;
}
// [TNLP_get_bounds_info]

// [TNLP_get_starting_point]
// returns the initial point for the problem
bool armtd_NLP::get_starting_point(
    Index   n,
    bool    init_x,
    Number* x,
    bool    init_z,
    Number* z_L,
    Number* z_U,
    Index   m,
    bool    init_lambda,
    Number* lambda
)
{
    // Here, we assume we only have starting values for x, if you code
    // your own NLP, you can provide starting values for the dual variables
    // if you wish
    if(init_x == false || init_z == true || init_lambda == true){
        WARNING_PRINT("*** Error wrong value of init in get_starting_point!");
    }

    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in get_starting_point!");
    }

    for( Index i = 0; i < n; i++ ) {
        // initialize to zero
        // x[i] = 0.0;

        // try to avoid local minimum
        x[i] = min(max((q_des[i] - desired_trajectory->q0[i]) / k_range[i], -0.5), 0.5);
    }

    return true;
}
// [TNLP_get_starting_point]

// [TNLP_eval_f]
// returns the value of the objective function
bool armtd_NLP::eval_f(
   Index         n,
   const Number* x,
   bool          new_x,
   Number&       obj_value
)
{
    if(n != NUM_FACTORS){
       WARNING_PRINT("*** Error wrong value of n in eval_f!");
    }

    // obj_value = sum((q_plan - q_des).^2);
    obj_value = 0; 
    for(Index i = 0; i < n; i++){
        TYPE q_plan = q_des_func(desired_trajectory->q0[i], desired_trajectory->qd0[i], desired_trajectory->qdd0[i], k_range[i] * x[i], t_plan);
        obj_value += pow(q_plan - q_des[i], 2);
    }

    obj_value *= 100.0;

    return true;
}
// [TNLP_eval_f]

// [TNLP_eval_grad_f]
// return the gradient of the objective function grad_{x} f(x)
bool armtd_NLP::eval_grad_f(
   Index         n,
   const Number* x,
   bool          new_x,
   Number*       grad_f
)
{
    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in eval_grad_f!");
    }

    for(Index i = 0; i < n; i++){
        TYPE q_plan = q_des_func(desired_trajectory->q0[i], desired_trajectory->qd0[i], desired_trajectory->qdd0[i], k_range[i] * x[i], t_plan);
        TYPE dk_q_plan = pow(t_plan,3) * (6 * pow(t_plan,2) - 15 * t_plan + 10);
        grad_f[i] = (2 * (q_plan - q_des[i]) * dk_q_plan * k_range[i]) * 100.0;
    }

    return true;
}
// [TNLP_eval_grad_f]

// [TNLP_eval_g]
// return the value of the constraints: g(x)
bool armtd_NLP::eval_g(
   Index         n,
   const Number* x,
   bool          new_x,
   Index         m,
   Number*       g
)
{
    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in eval_g!");
    }
    if(m != constraint_number){
        WARNING_PRINT("*** Error wrong value of m in eval_g!");
    }

    Index i;
    #pragma omp parallel for private(i) schedule(static, NUM_TIME_STEPS * NUM_FACTORS / NUM_THREADS)
    for(i = 0; i < NUM_TIME_STEPS * NUM_FACTORS; i++) {
        // Part 1. slice the control input PZ to get the center of the input bound, 
        //         while the radius of the input bound has already been incorporated in ipopt constraint bounds
        g[i] = getCenter(control_input[i].slice(x));

        // Part 2. slice the forward kinematics PZ to get the center of the joint position bound, 
        //         while the radius of the joint position bound has already been considered in Obstacles class
        checkJointsPosition[i * 3    ] = getCenter(joint_position[i * 3    ].slice(x));
        checkJointsPosition[i * 3 + 1] = getCenter(joint_position[i * 3 + 1].slice(x));
        checkJointsPosition[i * 3 + 2] = getCenter(joint_position[i * 3 + 2].slice(x));
    }

    // Part 3. check collision between joint position reachable set and obstacles (in gpu)
    obstacles->linkFRSConstraints(checkJointsPosition, nullptr, g + NUM_TIME_STEPS * NUM_FACTORS, nullptr);

    // Part 4. (position & velocity) state limit constraints
    desired_trajectory->returnJointPositionExtremum(g + NUM_TIME_STEPS * NUM_FACTORS + (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles, x);
    desired_trajectory->returnJointVelocityExtremum(g + NUM_TIME_STEPS * NUM_FACTORS + (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles + NUM_FACTORS * 2, x);

    // Part 5. force constraints on contact joint between tray and object
    #pragma omp parallel for private(i) schedule(static, NUM_TIME_STEPS*3 / NUM_THREADS)
    for(i = 0; i < NUM_TIME_STEPS*3; i++) {
        // put everything for force closure in here? or three for loops for each
    }
    // extract components of force
    f_c_x = f_c.elt[0];
    f_c_y = f_c.elt[1];
    f_c_z = f_c.elt[2];
    // extract components of moment
    n_c_x = n_c.elt[0]
    n_c_y = n_c.elt[1]
    n_c_z = n_c.elt[2]

    // separation constraint: -inf < -1*f_c_z < 0
    sep_value = getCenter(f_c_z.slice(x));
    // not sure what index to use here?
    g[NUM_TIME_STEPS * NUM_FACTORS + (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles + NUM_FACTORS * 2] = -1*sep_value;

    // need the center of the constraints (radius is used to buffer lower and upper bound)
    // have separate function in some appropriate file which calculates the constraint PZs?

    // all of the below could be called in armour_main.cpp and be a function elsewhere?
    // should have access to f,n here
    // split into components

    // in this file I need to slice and then calculate the constraints
    // slice
    // calculate constraints
    // take center of those constraints
    // pull out the f,n components here
    //  need to change call to rnea above to include the f,n
    //  also need to preallocate them

    // note: need to preallocate these for each time step
    

    return true;
}
// [TNLP_eval_g]


// [TNLP_eval_jac_g]
// return the structure or values of the Jacobian
bool armtd_NLP::eval_jac_g(
   Index         n,
   const Number* x,
   bool          new_x,
   Index         m,
   Index         nele_jac,
   Index*        iRow,
   Index*        jCol,
   Number*       values
)
{
    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in eval_g!");
    }
    if(m != constraint_number){
        WARNING_PRINT("*** Error wrong value of m in eval_g!");
    }
        
    if( values == NULL ) {
       // return the structure of the Jacobian
       // this particular Jacobian is dense
        for(Index i = 0; i < m; i++){
            for(Index j = 0; j < n; j++){
                iRow[i * n + j] = i;
                jCol[i * n + j] = j;
            }
        }
    }
    else {
        Index i;
        #pragma omp parallel for private(i) schedule(static, NUM_TIME_STEPS * NUM_FACTORS / NUM_THREADS)
        for(i = 0; i < NUM_TIME_STEPS * NUM_FACTORS; i++) {
            // Part 1. slice the control input PZ to get the center of the input bound, 
            //         while the radius of the input bound has already been incorporated in ipopt constraint bounds
            control_input[i].slice(values + i * NUM_FACTORS, x);

            // Part 2. slice the forward kinematics PZ to get the center of the joint position bound, 
            //         while the radius of the joint position bound has already been considered in Obstacles class
            checkJointsPosition[i * 3    ] = getCenter(joint_position[i * 3    ].slice(x));
            checkJointsPosition[i * 3 + 1] = getCenter(joint_position[i * 3 + 1].slice(x));
            checkJointsPosition[i * 3 + 2] = getCenter(joint_position[i * 3 + 2].slice(x));
            joint_position[i * 3    ].slice(dk_checkJointsPosition + (i * 3    ) * NUM_FACTORS, x);
            joint_position[i * 3 + 1].slice(dk_checkJointsPosition + (i * 3 + 1) * NUM_FACTORS, x);
            joint_position[i * 3 + 2].slice(dk_checkJointsPosition + (i * 3 + 2) * NUM_FACTORS, x);
        }

        // Part 3. check collision between joint position reachable set and obstacles (in gpu)
        obstacles->linkFRSConstraints(checkJointsPosition, dk_checkJointsPosition, nullptr, values + NUM_TIME_STEPS * NUM_FACTORS * NUM_FACTORS);

        // Part 4. (position & velocity) state limit constraints
        desired_trajectory->returnJointPositionExtremumGradient(values + (NUM_TIME_STEPS * NUM_FACTORS + (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles) * NUM_FACTORS, x);
        desired_trajectory->returnJointVelocityExtremumGradient(values + (NUM_TIME_STEPS * NUM_FACTORS + (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles + NUM_FACTORS * 2) * NUM_FACTORS, x);
    }

    return true;
}
// [TNLP_eval_jac_g]


// [TNLP_eval_h]
//return the structure or values of the Hessian
bool armtd_NLP::eval_h(
   Index         n,
   const Number* x,
   bool          new_x,
   Number        obj_factor,
   Index         m,
   const Number* lambda,
   bool          new_lambda,
   Index         nele_hess,
   Index*        iRow,
   Index*        jCol,
   Number*       values
)
{
    return false;
}
// [TNLP_eval_h]


// [TNLP_finalize_solution]
void armtd_NLP::finalize_solution(
    SolverReturn               status,
    Index                      n,
    const Number*              x,
    const Number*              z_L,
    const Number*              z_U,
    Index                      m,
    const Number*              g,
    const Number*              lambda,
    Number                     obj_value,
    const IpoptData*           ip_data,
    IpoptCalculatedQuantities* ip_cq
)
{
    // here is where we would store the solution to variables, or write to a file, etc
    // so we could use the solution.

    // store the solution
    for( Index i = 0; i < n; i++ ) {
        solution[i] = (TYPE)x[i];
    }

    // check constraint violation manually for Maximum_CpuTime_Exceeded case
    memcpy(g_copy, g, m * sizeof(Number));

    feasible = true;

    // control input constraints
    for( Index i = 0; i < NUM_TIME_STEPS; i++ ) {
        for( Index j = 0; j < NUM_FACTORS; j++ ) {
            if (g_copy[i * NUM_FACTORS + j] < -torque_limits[j] + v_norm[i * NUM_FACTORS + j] - TORQUE_INPUT_CONSTRAINT_VIOLATION_THRESHOLD || 
                g_copy[i * NUM_FACTORS + j] > torque_limits[j] - v_norm[i * NUM_FACTORS + j] + TORQUE_INPUT_CONSTRAINT_VIOLATION_THRESHOLD) {
                feasible = false;
                cout << "        CUDA & C++: Ipopt: Control torque of joint " << j << " at time interval " << i << " exceeds limit!\n";
                cout << "                        value: " << g_copy[i * NUM_FACTORS + j] << "\n";
                cout << "                        range: [ " << -torque_limits[j] + v_norm[i * NUM_FACTORS + j] << ", "
                                                            << torque_limits[j] - v_norm[i * NUM_FACTORS + j] << " ]\n";
                return;
            }
        }
    }    

    // collision avoidance constraints
    Index offset = NUM_FACTORS * NUM_TIME_STEPS;
    for( Index i = 0; i < NUM_FACTORS - 1; i++ ) {
        for( Index j = 0; j < NUM_TIME_STEPS; j++ ) {
            for( Index h = 0; h < obstacles->num_obstacles; h++ ) {
                if (g_copy[(i * NUM_TIME_STEPS + j) * obstacles->num_obstacles + h + offset] > COLLISION_AVOIDANCE_CONSTRAINT_VIOLATION_THRESHOLD) {
                    feasible = false;
                    cout << "        CUDA & C++: Ipopt: Collision between link " << i + 1 << " and obstacle " << h << " at time interval " << j << "!\n";
                    cout << "                        value: " << g_copy[(i * NUM_TIME_STEPS + j) * obstacles->num_obstacles + h + offset] << "\n";
                    return;
                }
            }
        }
    }
    offset += (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles;

    // state limit constraints
    //     minimum joint position
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        if (g_copy[i] < state_limits_lb[i - offset] + qe || g_copy[i] > state_limits_ub[i - offset] - qe) {
            feasible = false;
            cout << "        CUDA & C++: Ipopt: joint " << i - offset << " exceeds position limit when it reaches minimum!\n";
            cout << "                        value: " << g_copy[i] << "\n";
            cout << "                        range: [ " << state_limits_lb[i - offset] + qe << ", "
                                                        << state_limits_ub[i - offset] - qe << " ]\n";
            return;
        }
    }
    offset += NUM_FACTORS;

    //     maximum joint position
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        if (g_copy[i] < state_limits_lb[i - offset] + qe || g_copy[i] > state_limits_ub[i - offset] - qe) {
            feasible = false;
            cout << "        CUDA & C++: Ipopt: joint " << i - offset << " exceeds position limit when it reaches maximum!\n";
            cout << "                        value: " << g_copy[i] << "\n";
            cout << "                        range: [ " << state_limits_lb[i - offset] + qe << ", "
                                                        << state_limits_ub[i - offset] - qe << " ]\n";
            return;
        }
    }
    offset += NUM_FACTORS;

    //     minimum joint velocity
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        if (g_copy[i] < -speed_limits[i - offset] + qde || g_copy[i] > speed_limits[i - offset] - qde) {
            feasible = false;
            cout << "        CUDA & C++: Ipopt: joint " << i - offset << " exceeds velocity limit when it reaches minimum!\n";
            cout << "                        value: " << g_copy[i] << "\n";
            cout << "                        range: [ " << -speed_limits[i - offset] + qde << ", "
                                                        << speed_limits[i - offset] - qde << " ]\n";
            return;
        }
    }
    offset += NUM_FACTORS;

    //     maximum joint velocity
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        if (g_copy[i] < -speed_limits[i - offset] + qde || g_copy[i] > speed_limits[i - offset] - qde) {
            feasible = false;
            cout << "        CUDA & C++: Ipopt: joint " << i - offset << " exceeds velocity limit when it reaches maximum!\n";
            cout << "                        value: " << g_copy[i] << "\n";
            cout << "                        range: [ " << -speed_limits[i - offset] + qde << ", "
                                                        << speed_limits[i - offset] - qde << " ]\n";
            return;
        }
    }
}
// [TNLP_finalize_solution]


#endif
