#include "hip/hip_runtime.h"
#ifndef NLP_CLASS_CU
#define NLP_CLASS_CU

#include "NLPclass.h"

// constructor
armtd_NLP::armtd_NLP()
{
    checkJointsPosition = new TYPE[NUM_TIME_STEPS * NUM_FACTORS * 3];
    dk_checkJointsPosition = new TYPE[NUM_TIME_STEPS * NUM_FACTORS * 3 * NUM_FACTORS];
}


// destructor
armtd_NLP::~armtd_NLP()
{
    delete[] checkJointsPosition;
    delete[] dk_checkJointsPosition;
    delete[] g_copy;
}


bool armtd_NLP::set_parameters(
    TYPE* q_des_input,
    TYPE t_plan_input,
    BezierCurve* desired_trajectory_input,
    PZsparse* joint_position_input,
    PZsparse* control_input_input,
    TYPE* v_norm_input,
    Obstacles* obstacles_input,
    vecPZsparse* f_c_input,
    vecPZsparse* n_c_input,
    const Number* u_s
 ) 
 {
    q_des = q_des_input;
    t_plan = t_plan_input;
    desired_trajectory = desired_trajectory_input;
    joint_position = joint_position_input;
    control_input = control_input_input;
    v_norm = v_norm_input;
    obstacles = obstacles_input;
    f_c = f_c_input;
    n_c = n_c_input;

    constraint_number = NUM_FACTORS * NUM_TIME_STEPS +
                        (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles + 
                        NUM_FACTORS * 4 + 
                        NUM_TIME_STEPS * 3;
    // 1. torque constraints
    // 2. collision checking constraints (ignoring base link and no self-collision)
    // 3. joint limit constraints, pos/vel lower and upper constraints
    // 4. force constraints

    g_copy = new Number[constraint_number];

    return true;
}


bool armtd_NLP::get_nlp_info(
   Index&          n,
   Index&          m,
   Index&          nnz_jac_g,
   Index&          nnz_h_lag,
   IndexStyleEnum& index_style
)
{
    // The problem described NUM_FACTORS variables, x[NUM_FACTORS] through x[NUM_FACTORS] for each joint
    n = NUM_FACTORS;

    // number of inequality constraint
    m = constraint_number;

    nnz_jac_g = m * n;

    // use the C style indexing (0-based)
    index_style = TNLP::C_STYLE;

    return true;
}
// [TNLP_get_nlp_info]

// [TNLP_get_bounds_info]
// returns the variable bounds
bool armtd_NLP::get_bounds_info(
   Index   n,
   Number* x_l,
   Number* x_u,
   Index   m,
   Number* g_l,
   Number* g_u
)
{
    // here, the n and m we gave IPOPT in get_nlp_info are passed back to us.
    // If desired, we could assert to make sure they are what we think they are.
    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in get_bounds_info!");
    }
    if(m != constraint_number){
        WARNING_PRINT("*** Error wrong value of m in get_bounds_info!");
    }

    // lower bounds
    for( Index i = 0; i < n; i++ ) {
        x_l[i] = -1.0;
    }

    // upper bounds  
    for( Index i = 0; i < n; i++ ) {
        x_u[i] = 1.0;
    }

    // control input constraints
    for( Index i = 0; i < NUM_TIME_STEPS; i++ ) {
        for( Index j = 0; j < NUM_FACTORS; j++ ) {
            g_l[i * NUM_FACTORS + j] = -torque_limits[j] + v_norm[i * NUM_FACTORS + j];
            g_u[i * NUM_FACTORS + j] = torque_limits[j] - v_norm[i * NUM_FACTORS + j];
        }
    }    
    Index offset = NUM_FACTORS * NUM_TIME_STEPS;

    // collision avoidance constraints
    for( Index i = offset; i < offset + (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles; i++ ) {
        g_l[i] = -1e19;
        g_u[i] = 0;
    }
    offset += (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles;

    // state limit constraints
    //     minimum joint position
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        g_l[i] = state_limits_lb[i - offset] + qe;
        g_u[i] = state_limits_ub[i - offset] - qe;
    }
    offset += NUM_FACTORS;

    //     maximum joint position
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        g_l[i] = state_limits_lb[i - offset] + qe;
        g_u[i] = state_limits_ub[i - offset] - qe;
    }
    offset += NUM_FACTORS;

    //     minimum joint velocity
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        g_l[i] = -speed_limits[i - offset] + qde;
        g_u[i] = speed_limits[i - offset] - qde;
    }
    offset += NUM_FACTORS;

    //     maximum joint velocity
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        g_l[i] = -speed_limits[i - offset] + qde;
        g_u[i] = speed_limits[i - offset] - qde;
    }
    offset += NUM_FACTORS;



    // force constraints on contact joint between tray and cup
    // need to add the radius of the constraints here (like how is done with the torque constraints, v_norm passes out the radius I think)
    // double check signs and how the torque constraints are buffered using the radius
    // for v_norm in armour_main.cpp : getRadius(u_nom[t_ind * NUM_FACTORS + i].independent)


    //     separation constraint
    // upper bound should be zero and lower bound should be -inf
    for( Index i = offset; i < offset + NUM_TIME_STEPS; i++){
        g_l[i] = -1e19;
        g_u[i] = 0;
    }
    offset += NUM_TIME_STEPS;

    //     slipping constraint
    // upper bound should be zero and lower bound should be -inf for the reformulated constraint (not the normal friction law?)
    for( Index i = offset; i < offset + NUM_TIME_STEPS; i++){
        g_l[i] = -1e19;
        g_u[i] = 0;
    }
    offset += NUM_TIME_STEPS;

    //     tipping constraint
    // upper bound should be zero and lower bound should be -inf for the reformulated constraint
    for( Index i = offset; i < offset + NUM_TIME_STEPS; i++){
        g_l[i] = -1e19;
        g_u[i] = 0;
    }
    offset += NUM_TIME_STEPS;

    return true;
}
// [TNLP_get_bounds_info]

// [TNLP_get_starting_point]
// returns the initial point for the problem
bool armtd_NLP::get_starting_point(
    Index   n,
    bool    init_x,
    Number* x,
    bool    init_z,
    Number* z_L,
    Number* z_U,
    Index   m,
    bool    init_lambda,
    Number* lambda
)
{
    // Here, we assume we only have starting values for x, if you code
    // your own NLP, you can provide starting values for the dual variables
    // if you wish
    if(init_x == false || init_z == true || init_lambda == true){
        WARNING_PRINT("*** Error wrong value of init in get_starting_point!");
    }

    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in get_starting_point!");
    }

    for( Index i = 0; i < n; i++ ) {
        // initialize to zero
        // x[i] = 0.0;

        // try to avoid local minimum
        x[i] = min(max((q_des[i] - desired_trajectory->q0[i]) / k_range[i], -0.5), 0.5);
    }

    return true;
}
// [TNLP_get_starting_point]

// [TNLP_eval_f]
// returns the value of the objective function
bool armtd_NLP::eval_f(
   Index         n,
   const Number* x,
   bool          new_x,
   Number&       obj_value
)
{
    if(n != NUM_FACTORS){
       WARNING_PRINT("*** Error wrong value of n in eval_f!");
    }

    // obj_value = sum((q_plan - q_des).^2);
    obj_value = 0; 
    for(Index i = 0; i < n; i++){
        TYPE q_plan = q_des_func(desired_trajectory->q0[i], desired_trajectory->qd0[i], desired_trajectory->qdd0[i], k_range[i] * x[i], t_plan);
        obj_value += pow(q_plan - q_des[i], 2);
    }

    obj_value *= 100.0;

    return true;
}
// [TNLP_eval_f]

// [TNLP_eval_grad_f]
// return the gradient of the objective function grad_{x} f(x)
bool armtd_NLP::eval_grad_f(
   Index         n,
   const Number* x,
   bool          new_x,
   Number*       grad_f
)
{
    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in eval_grad_f!");
    }

    for(Index i = 0; i < n; i++){
        TYPE q_plan = q_des_func(desired_trajectory->q0[i], desired_trajectory->qd0[i], desired_trajectory->qdd0[i], k_range[i] * x[i], t_plan);
        TYPE dk_q_plan = pow(t_plan,3) * (6 * pow(t_plan,2) - 15 * t_plan + 10);
        grad_f[i] = (2 * (q_plan - q_des[i]) * dk_q_plan * k_range[i]) * 100.0;
    }

    return true;
}
// [TNLP_eval_grad_f]

// [TNLP_eval_g]
// return the value of the constraints: g(x)
bool armtd_NLP::eval_g(
   Index         n,
   const Number* x,
   bool          new_x,
   Index         m,
   Number*       g
)
{
    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in eval_g!");
    }
    if(m != constraint_number){
        WARNING_PRINT("*** Error wrong value of m in eval_g!");
    }

    Index i;
    #pragma omp parallel for private(i) schedule(static, NUM_TIME_STEPS * NUM_FACTORS / NUM_THREADS)
    for(i = 0; i < (NUM_TIME_STEPS * NUM_FACTORS + NUM_TIME_STEPS*3); i++) {
        
        if(i < (NUM_TIME_STEPS * NUM_FACTORS)) {
            // Part 1. slice the control input PZ to get the center of the input bound, 
            //         while the radius of the input bound has already been incorporated in ipopt constraint bounds
            g[i] = getCenter(control_input[i].slice(x));

            // Part 2. slice the forward kinematics PZ to get the center of the joint position bound, 
            //         while the radius of the joint position bound has already been considered in Obstacles class
            checkJointsPosition[i * 3    ] = getCenter(joint_position[i * 3    ].slice(x));
            checkJointsPosition[i * 3 + 1] = getCenter(joint_position[i * 3 + 1].slice(x));
            checkJointsPosition[i * 3 + 2] = getCenter(joint_position[i * 3 + 2].slice(x));
        } 
        else {
            // Part 3. force constraints on contact joint between tray and object

            // extract components of force (is f_c and n_c a single time step here or an array of all of them?)
            PZsparse* f_c_x = f_c[i].elt[0];
            PZsparse* f_c_y = f_c[i].elt[1];
            PZsparse* f_c_z = f_c[i].elt[2];
            // get centers
            Number* f_c_x_center = getCenter(f_c_x);
            Number* f_c_y_center = getCenter(f_c_y);
            Number* f_c_z_center = getCenter(f_c_z);
            // get radii of independent generators
            TYPE* f_c_x_radius = getRadius(f_c_x.independent)
            TYPE* f_c_y_radius = getRadius(f_c_y.independent)
            TYPE* f_c_z_radius = getRadius(f_c_z.independent)

            // extract components of moment
            PZsparse* n_c_x = n_c[i].elt[0]
            PZsparse* n_c_y = n_c[i].elt[1]
            PZsparse* n_c_z = n_c[i].elt[2]

            // separation constraint: -inf < -1*f_c_z < 0
            // storing separation constraint value, not sure what index to use here?
            g[i] = -1*getCenter(f_c_z.slice(x));

            // slipping constraint: -inf < f_c_x*f_c_x + f_c_y*f_c_y - u_s^2*f_c_z*f_c_z < 0
            // need to write this constraint differently than matlab implementation as we need to 
            // slice before doing multiplications of PZs in order to avoid memory problems.
            // getCenter, getRadius (for v_norm in armour_main.cpp : getRadius(u_nom[t_ind * NUM_FACTORS + i].independent))
            
            // check the signs of the centers of the force zonotopes
            // condition 1: all positive
            if ( (f_c_x_center >= 0) && (f_c_y_center >= 0) && (f_c_z_center >= 0) ){
                // Note: double check that the center/radius is a number that can be squared
                g[i+NUM_TIME_STEPS] = f_c_x_center^2 + 2*f_c_x_radius*f_c_x_center + f_c_x_radius^2 + f_c_y_center^2 + 2*f_c_y_radius*f_c_y_center + f_c_y_radius^2 - u_s^2 * ( f_c_z_center^2 - 2*f_c_z_radius*f_c_z_center - f_c_z_radius^2);
            }
            // condition 2: y negative
            else if ( (f_c_x_center >= 0) && (f_c_y_center <= 0) && (f_c_z_center >= 0) ) {
                g[i+NUM_TIME_STEPS] = f_c_x_center^2 + 2*f_c_x_radius*f_c_x_center + f_c_x_radius^2 + f_c_y_center^2 + 2*f_c_y_radius*f_c_y_center + f_c_y_radius^2 - u_s^2 * ( f_c_z_center^2 - 2*f_c_z_radius*f_c_z_center - f_c_z_radius^2);
            }
            // condition 3: z negative
            else if ( (f_c_x_center >= 0) && (f_c_y_center >= 0) && (f_c_z_center <= 0) ) {
                g[i+NUM_TIME_STEPS] = f_c_x_center^2 + 2*f_c_x_radius*f_c_x_center + f_c_x_radius^2 + f_c_y_center^2 + 2*f_c_y_radius*f_c_y_center + f_c_y_radius^2 - u_s^2 * ( f_c_z_center^2 - 2*f_c_z_radius*f_c_z_center - f_c_z_radius^2);
            }
            // condition 4: y and z negative
            else if ( (f_c_x_center >= 0) && (f_c_y_center <= 0) && (f_c_z_center <= 0) ) {
                g[i+NUM_TIME_STEPS] = f_c_x_center^2 + 2*f_c_x_radius*f_c_x_center + f_c_x_radius^2 + f_c_y_center^2 + 2*f_c_y_radius*f_c_y_center + f_c_y_radius^2 - u_s^2 * ( f_c_z_center^2 - 2*f_c_z_radius*f_c_z_center - f_c_z_radius^2);
            }
            // condition 5: x negative
            else if ( (f_c_x_center <= 0) && (f_c_y_center >= 0) && (f_c_z_center >= 0) ) {
                g[i+NUM_TIME_STEPS] = f_c_x_center^2 + 2*f_c_x_radius*f_c_x_center + f_c_x_radius^2 + f_c_y_center^2 + 2*f_c_y_radius*f_c_y_center + f_c_y_radius^2 - u_s^2 * ( f_c_z_center^2 - 2*f_c_z_radius*f_c_z_center - f_c_z_radius^2);
            }
            // condition 6: x and y negative
            else if ( (f_c_x_center <= 0) && (f_c_y_center <= 0) && (f_c_z_center >= 0) ) {
                g[i+NUM_TIME_STEPS] = f_c_x_center^2 + 2*f_c_x_radius*f_c_x_center + f_c_x_radius^2 + f_c_y_center^2 + 2*f_c_y_radius*f_c_y_center + f_c_y_radius^2 - u_s^2 * ( f_c_z_center^2 - 2*f_c_z_radius*f_c_z_center - f_c_z_radius^2);
            }
            // condition 7: x and z negative
            else if ( (f_c_x_center <= 0) && (f_c_y_center >= 0) && (f_c_z_center <= 0) ) {
                g[i+NUM_TIME_STEPS] = f_c_x_center^2 + 2*f_c_x_radius*f_c_x_center + f_c_x_radius^2 + f_c_y_center^2 + 2*f_c_y_radius*f_c_y_center + f_c_y_radius^2 - u_s^2 * ( f_c_z_center^2 - 2*f_c_z_radius*f_c_z_center - f_c_z_radius^2);
            }
            // condition 8: x and y and z negative
            else if ( (f_c_x_center <= 0) && (f_c_y_center <= 0) && (f_c_z_center <= 0) ) {
                g[i+NUM_TIME_STEPS] = f_c_x_center^2 + 2*f_c_x_radius*f_c_x_center + f_c_x_radius^2 + f_c_y_center^2 + 2*f_c_y_radius*f_c_y_center + f_c_y_radius^2 - u_s^2 * ( f_c_z_center^2 - 2*f_c_z_radius*f_c_z_center - f_c_z_radius^2);
            }

            // storing tipping constraint value, not sure what index to use here?
            g[i+NUM_TIME_STEPS*2] = ;

        }
        
    }

    // Bohao says to put contact constraints into the same loop
    //  adjust the index length (+NUM_TIME_STEPS*3)
    //  separate joint position constraint and force constraints by using if/else statement to check index?
    //    if i < NUM_TIME_STEPS * NUM_FACTORS (0 -> NUM_TIME_STEPS*NUM_FACTORS)
    //      joint_position constraint
    //    else
    //      contact constraints (NUM_TIME_STEPS*NUM_FACTORS -> (NUM_TIME_STEPS*NUM_FACTORS+NUM_TIME_STEPS*3))
    //  adjust the indices of the following constraints

    // Part 4. check collision between joint position reachable set and obstacles (in gpu)
    obstacles->linkFRSConstraints(checkJointsPosition, nullptr, g + (NUM_TIME_STEPS * NUM_FACTORS + NUM_TIME_STEPS*3), nullptr);

    // Part 5. (position & velocity) state limit constraints
    desired_trajectory->returnJointPositionExtremum(g + (NUM_TIME_STEPS * NUM_FACTORS + NUM_TIME_STEPS*3) + (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles, x);
    desired_trajectory->returnJointVelocityExtremum(g + (NUM_TIME_STEPS * NUM_FACTORS + NUM_TIME_STEPS*3) + (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles + NUM_FACTORS * 2, x);

    // Part 5. force constraints on contact joint between tray and object
        // put everything for force closure in here? or three for loops for each

        
        // need the center of the constraints (radius is used to buffer lower and upper bound)
        // have separate function in some appropriate file which calculates the constraint PZs?

        // all of the below could be called in armour_main.cpp and be a function elsewhere?
        // should have access to f,n here
        // split into components

        // in this file I need to slice and then calculate the constraints
        // slice
        // calculate constraints
        // take center of those constraints
        // pull out the f,n components here
        //  need to change call to rnea above to include the f,n
        //  also need to preallocate them

        // note: need to preallocate these for each time step

    }

    

    return true;
}
// [TNLP_eval_g]


// [TNLP_eval_jac_g]
// return the structure or values of the Jacobian
bool armtd_NLP::eval_jac_g(
   Index         n,
   const Number* x,
   bool          new_x,
   Index         m,
   Index         nele_jac,
   Index*        iRow,
   Index*        jCol,
   Number*       values
)
{
    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in eval_g!");
    }
    if(m != constraint_number){
        WARNING_PRINT("*** Error wrong value of m in eval_g!");
    }
        
    if( values == NULL ) {
       // return the structure of the Jacobian
       // this particular Jacobian is dense
        for(Index i = 0; i < m; i++){
            for(Index j = 0; j < n; j++){
                iRow[i * n + j] = i;
                jCol[i * n + j] = j;
            }
        }
    }
    else {
        Index i;
        #pragma omp parallel for private(i) schedule(static, NUM_TIME_STEPS * NUM_FACTORS / NUM_THREADS)
        for(i = 0; i < NUM_TIME_STEPS * NUM_FACTORS; i++) {
            // Part 1. slice the control input PZ to get the center of the input bound, 
            //         while the radius of the input bound has already been incorporated in ipopt constraint bounds
            control_input[i].slice(values + i * NUM_FACTORS, x);

            // Part 2. slice the forward kinematics PZ to get the center of the joint position bound, 
            //         while the radius of the joint position bound has already been considered in Obstacles class
            checkJointsPosition[i * 3    ] = getCenter(joint_position[i * 3    ].slice(x));
            checkJointsPosition[i * 3 + 1] = getCenter(joint_position[i * 3 + 1].slice(x));
            checkJointsPosition[i * 3 + 2] = getCenter(joint_position[i * 3 + 2].slice(x));
            joint_position[i * 3    ].slice(dk_checkJointsPosition + (i * 3    ) * NUM_FACTORS, x);
            joint_position[i * 3 + 1].slice(dk_checkJointsPosition + (i * 3 + 1) * NUM_FACTORS, x);
            joint_position[i * 3 + 2].slice(dk_checkJointsPosition + (i * 3 + 2) * NUM_FACTORS, x);
        }

        // Part 3. check collision between joint position reachable set and obstacles (in gpu)
        obstacles->linkFRSConstraints(checkJointsPosition, dk_checkJointsPosition, nullptr, values + NUM_TIME_STEPS * NUM_FACTORS * NUM_FACTORS);

        // Part 4. (position & velocity) state limit constraints
        desired_trajectory->returnJointPositionExtremumGradient(values + (NUM_TIME_STEPS * NUM_FACTORS + (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles) * NUM_FACTORS, x);
        desired_trajectory->returnJointVelocityExtremumGradient(values + (NUM_TIME_STEPS * NUM_FACTORS + (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles + NUM_FACTORS * 2) * NUM_FACTORS, x);
    }

    return true;
}
// [TNLP_eval_jac_g]


// [TNLP_eval_h]
//return the structure or values of the Hessian
bool armtd_NLP::eval_h(
   Index         n,
   const Number* x,
   bool          new_x,
   Number        obj_factor,
   Index         m,
   const Number* lambda,
   bool          new_lambda,
   Index         nele_hess,
   Index*        iRow,
   Index*        jCol,
   Number*       values
)
{
    return false;
}
// [TNLP_eval_h]


// [TNLP_finalize_solution]
void armtd_NLP::finalize_solution(
    SolverReturn               status,
    Index                      n,
    const Number*              x,
    const Number*              z_L,
    const Number*              z_U,
    Index                      m,
    const Number*              g,
    const Number*              lambda,
    Number                     obj_value,
    const IpoptData*           ip_data,
    IpoptCalculatedQuantities* ip_cq
)
{
    // here is where we would store the solution to variables, or write to a file, etc
    // so we could use the solution.

    // store the solution
    for( Index i = 0; i < n; i++ ) {
        solution[i] = (TYPE)x[i];
    }

    // check constraint violation manually for Maximum_CpuTime_Exceeded case
    memcpy(g_copy, g, m * sizeof(Number));

    feasible = true;

    // control input constraints
    for( Index i = 0; i < NUM_TIME_STEPS; i++ ) {
        for( Index j = 0; j < NUM_FACTORS; j++ ) {
            if (g_copy[i * NUM_FACTORS + j] < -torque_limits[j] + v_norm[i * NUM_FACTORS + j] - TORQUE_INPUT_CONSTRAINT_VIOLATION_THRESHOLD || 
                g_copy[i * NUM_FACTORS + j] > torque_limits[j] - v_norm[i * NUM_FACTORS + j] + TORQUE_INPUT_CONSTRAINT_VIOLATION_THRESHOLD) {
                feasible = false;
                cout << "        CUDA & C++: Ipopt: Control torque of joint " << j << " at time interval " << i << " exceeds limit!\n";
                cout << "                        value: " << g_copy[i * NUM_FACTORS + j] << "\n";
                cout << "                        range: [ " << -torque_limits[j] + v_norm[i * NUM_FACTORS + j] << ", "
                                                            << torque_limits[j] - v_norm[i * NUM_FACTORS + j] << " ]\n";
                return;
            }
        }
    }    

    // collision avoidance constraints
    Index offset = NUM_FACTORS * NUM_TIME_STEPS;
    for( Index i = 0; i < NUM_FACTORS - 1; i++ ) {
        for( Index j = 0; j < NUM_TIME_STEPS; j++ ) {
            for( Index h = 0; h < obstacles->num_obstacles; h++ ) {
                if (g_copy[(i * NUM_TIME_STEPS + j) * obstacles->num_obstacles + h + offset] > COLLISION_AVOIDANCE_CONSTRAINT_VIOLATION_THRESHOLD) {
                    feasible = false;
                    cout << "        CUDA & C++: Ipopt: Collision between link " << i + 1 << " and obstacle " << h << " at time interval " << j << "!\n";
                    cout << "                        value: " << g_copy[(i * NUM_TIME_STEPS + j) * obstacles->num_obstacles + h + offset] << "\n";
                    return;
                }
            }
        }
    }
    offset += (NUM_FACTORS - 1) * NUM_TIME_STEPS * obstacles->num_obstacles;

    // state limit constraints
    //     minimum joint position
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        if (g_copy[i] < state_limits_lb[i - offset] + qe || g_copy[i] > state_limits_ub[i - offset] - qe) {
            feasible = false;
            cout << "        CUDA & C++: Ipopt: joint " << i - offset << " exceeds position limit when it reaches minimum!\n";
            cout << "                        value: " << g_copy[i] << "\n";
            cout << "                        range: [ " << state_limits_lb[i - offset] + qe << ", "
                                                        << state_limits_ub[i - offset] - qe << " ]\n";
            return;
        }
    }
    offset += NUM_FACTORS;

    //     maximum joint position
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        if (g_copy[i] < state_limits_lb[i - offset] + qe || g_copy[i] > state_limits_ub[i - offset] - qe) {
            feasible = false;
            cout << "        CUDA & C++: Ipopt: joint " << i - offset << " exceeds position limit when it reaches maximum!\n";
            cout << "                        value: " << g_copy[i] << "\n";
            cout << "                        range: [ " << state_limits_lb[i - offset] + qe << ", "
                                                        << state_limits_ub[i - offset] - qe << " ]\n";
            return;
        }
    }
    offset += NUM_FACTORS;

    //     minimum joint velocity
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        if (g_copy[i] < -speed_limits[i - offset] + qde || g_copy[i] > speed_limits[i - offset] - qde) {
            feasible = false;
            cout << "        CUDA & C++: Ipopt: joint " << i - offset << " exceeds velocity limit when it reaches minimum!\n";
            cout << "                        value: " << g_copy[i] << "\n";
            cout << "                        range: [ " << -speed_limits[i - offset] + qde << ", "
                                                        << speed_limits[i - offset] - qde << " ]\n";
            return;
        }
    }
    offset += NUM_FACTORS;

    //     maximum joint velocity
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        if (g_copy[i] < -speed_limits[i - offset] + qde || g_copy[i] > speed_limits[i - offset] - qde) {
            feasible = false;
            cout << "        CUDA & C++: Ipopt: joint " << i - offset << " exceeds velocity limit when it reaches maximum!\n";
            cout << "                        value: " << g_copy[i] << "\n";
            cout << "                        range: [ " << -speed_limits[i - offset] + qde << ", "
                                                        << speed_limits[i - offset] - qde << " ]\n";
            return;
        }
    }
}
// [TNLP_finalize_solution]


#endif
