#include "hip/hip_runtime.h"
#ifndef NLP_CLASS_CU
#define NLP_CLASS_CU
#include <iostream>

#include "NLPclass.h"

 
double wrap_to_pi(const double angle) {
    double wrapped_angle = angle;
    while (wrapped_angle < -M_PI) {
        wrapped_angle += 2*M_PI;
    }
    while (wrapped_angle > M_PI) {
        wrapped_angle -= 2*M_PI;
    }
    return wrapped_angle;
}

// constructor
armtd_NLP::armtd_NLP()
{
}


// destructor
armtd_NLP::~armtd_NLP()
{
    delete[] g_copy;
}


bool armtd_NLP::set_parameters(
    Eigen::VectorXd& q_des_input,
    double t_plan_input,
    const BezierCurve* desired_trajectory_input,
    KinematicsDynamics* kinematics_dynamics_result_input,
    const Eigen::MatrixXd* torque_radius_input,
    Obstacles* obstacles_input
 ) 
 {
    q_des = q_des_input;
    t_plan = t_plan_input;
    desired_trajectory = desired_trajectory_input;
    kinematics_dynamics_result = kinematics_dynamics_result_input;
    torque_radius = torque_radius_input;
    obstacles = obstacles_input;

    if (!TURN_OFF_INPUT_CONSTRAINTS) {
        constraint_number = NUM_FACTORS * NUM_TIME_STEPS +
                            NUM_JOINTS * NUM_TIME_STEPS * obstacles->num_obstacles + 
                            NUM_FACTORS * 4;
    }
    else {
        constraint_number = NUM_JOINTS * NUM_TIME_STEPS * obstacles->num_obstacles + 
                            NUM_FACTORS * 4;
    }

    g_copy = new Number[constraint_number];

    return true;
}


bool armtd_NLP::get_nlp_info(
   Index&          n,
   Index&          m,
   Index&          nnz_jac_g,
   Index&          nnz_h_lag,
   IndexStyleEnum& index_style
)
{
    // The problem described NUM_FACTORS variables, x[NUM_FACTORS] through x[NUM_FACTORS] for each joint
    n = NUM_FACTORS;

    // number of inequality constraint
    m = constraint_number;

    nnz_jac_g = m * n;

    // use the C style indexing (0-based)
    index_style = TNLP::C_STYLE;

    return true;
}
// [TNLP_get_nlp_info]

// [TNLP_get_bounds_info]
// returns the variable bounds
bool armtd_NLP::get_bounds_info(
   Index   n,
   Number* x_l,
   Number* x_u,
   Index   m,
   Number* g_l,
   Number* g_u
)
{
    // here, the n and m we gave IPOPT in get_nlp_info are passed back to us.
    // If desired, we could assert to make sure they are what we think they are.
    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in get_bounds_info!");
    }
    if(m != constraint_number){
        WARNING_PRINT("*** Error wrong value of m in get_bounds_info!");
    }

    // lower bounds
    for( Index i = 0; i < n; i++ ) {
        x_l[i] = -1.0;
    }

    // upper bounds  
    for( Index i = 0; i < n; i++ ) {
        x_u[i] = 1.0;
    }

    Index offset = 0;

    if (!TURN_OFF_INPUT_CONSTRAINTS) {
        // control input constraints
        for( Index i = 0; i < NUM_TIME_STEPS; i++ ) {
            for( Index j = 0; j < NUM_FACTORS; j++ ) {
                g_l[i * NUM_FACTORS + j] = -torque_limits[j] + (*torque_radius)(j, i);
                g_u[i * NUM_FACTORS + j] = torque_limits[j] - (*torque_radius)(j, i);
            }
        }    

        offset += NUM_FACTORS * NUM_TIME_STEPS;
    }

    // collision avoidance constraints
    for( Index i = offset; i < offset + NUM_TIME_STEPS * NUM_JOINTS * obstacles->num_obstacles; i++ ) {
        g_l[i] = -1e19;
        g_u[i] = 0;
    }
    offset += NUM_TIME_STEPS * NUM_JOINTS * obstacles->num_obstacles;

    // state limit constraints
    //     minimum joint position
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        g_l[i] = state_limits_lb[i - offset] + qe;
        g_u[i] = state_limits_ub[i - offset] - qe;
    }
    offset += NUM_FACTORS;

    //     maximum joint position
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        g_l[i] = state_limits_lb[i - offset] + qe;
        g_u[i] = state_limits_ub[i - offset] - qe;
    }
    offset += NUM_FACTORS;

    //     minimum joint velocity
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        g_l[i] = -speed_limits[i - offset] + qde;
        g_u[i] = speed_limits[i - offset] - qde;
    }
    offset += NUM_FACTORS;

    //     maximum joint velocity
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        g_l[i] = -speed_limits[i - offset] + qde;
        g_u[i] = speed_limits[i - offset] - qde;
    }

    return true;
}
// [TNLP_get_bounds_info]

// [TNLP_get_starting_point]
// returns the initial point for the problem
bool armtd_NLP::get_starting_point(
    Index   n,
    bool    init_x,
    Number* x,
    bool    init_z,
    Number* z_L,
    Number* z_U,
    Index   m,
    bool    init_lambda,
    Number* lambda
)
{
    // Here, we assume we only have starting values for x, if you code
    // your own NLP, you can provide starting values for the dual variables
    // if you wish
    if(init_x == false || init_z == true || init_lambda == true){
        WARNING_PRINT("*** Error wrong value of init in get_starting_point!");
    }

    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in get_starting_point!");
    }

    for( Index i = 0; i < n; i++ ) {
        // initialize to zero
        x[i] = 0.0;

        // try to avoid local minimum
        // x[i] = min(max((q_des[i] - dg_copyesired_trajectory->q0[i]) / k_range[i], -0.5), 0.5);
    }

    return true;
}
// [TNLP_get_starting_point]

// [TNLP_eval_f]
// returns the value of the objective function
bool armtd_NLP::eval_f(
   Index         n,
   const Number* x,
   bool          new_x,
   Number&       obj_value
)
{
    if(n != NUM_FACTORS){
       WARNING_PRINT("*** Error wrong value of n in eval_f!");
    }

    // obj_value = sum((q_plan - q_des).^2);
    Eigen::VectorXd q_plan(n);
    for(Index i = 0; i < n; i++){
        q_plan[i] = q_des_func(desired_trajectory->q0[i], desired_trajectory->Tqd0[i], desired_trajectory->TTqdd0[i], k_range[i] * x[i], t_plan);
  //normal      //q_plan[i] = q_des_func(desired_trajectory->q0[i], desired_trajectory->Tqd0[i], desired_trajectory->TTqdd0[i], k_range[i] * x[i], t_plan);
    }

    // kinova has 4 infinite rotation joints
    // //Normal approach
    // obj_value = pow(wrap_to_pi(q_des[0] - q_plan[0]), 2) +
    //             pow(wrap_to_pi(q_des[2] - q_plan[2]), 2) + 
    //             pow(wrap_to_pi(q_des[4] - q_plan[4]), 2) + 
    //             pow(wrap_to_pi(q_des[6] - q_plan[6]), 2) + 
    //             pow(q_des[1] - q_plan[1], 2) + 
    //             pow(q_des[3] - q_plan[3], 2) + 
    //             pow(q_des[5] - q_plan[5], 2);
    obj_value = -q_plan[6];
    // sqrt(q_plan[0]*q_plan[0] +
    // q_plan[1]*q_plan[1]+
    // q_plan[2]*q_plan[2]+
    // q_plan[3]*q_plan[3]+
    // q_plan[4]*q_plan[4]+
    // q_plan[5]*q_plan[5] +
    // q_plan[6]*q_plan[6]);

    obj_value *= COST_FUNCTION_OPTIMALITY_SCALE;

    return true;
}
// [TNLP_eval_f]

// [TNLP_eval_grad_f]
// return the gradient of the objective function grad_{x} f(x)
bool armtd_NLP::eval_grad_f(
   Index         n,
   const Number* x,
   bool          new_x,
   Number*       grad_f
)
{
    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in eval_grad_f!");
    }

    for(Index i = 0; i < n; i++){
        double q_plan = q_des_func(desired_trajectory->q0[i], desired_trajectory->Tqd0[i], desired_trajectory->TTqdd0[i], k_range[i] * x[i], t_plan);
        double dk_q_plan = pow(t_plan,3) * (6 * pow(t_plan,2) - 15 * t_plan + 10) * k_range[i];

        // // kinova has 4 infinite rotation joints
        // if (i % 2 == 0) {
        //     grad_f[i] = (2 * wrap_to_pi(q_plan - q_des[i]) * dk_q_plan);
        // }
        // else {
        //     grad_f[i] = (2 * (q_plan - q_des[i]) * dk_q_plan);
        // }


    }
        grad_f[6] = -qd_deriv_function(desired_trajectory->q0[6], desired_trajectory->Tqd0[6], desired_trajectory->TTqdd0[6], k_range[6] * x[6], t_plan)*k_range[6];
        grad_f[6] *= COST_FUNCTION_OPTIMALITY_SCALE;


    return true;
}
// [TNLP_eval_grad_f]

// [TNLP_eval_g]
// return the value of the constraints: g(x)
bool armtd_NLP::eval_g(
   Index         n,
   const Number* x,
   bool          new_x,
   Index         m,
   Number*       g
)
{
    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in eval_g!");
    }
    if(m != constraint_number){
        WARNING_PRINT("*** Error wrong value of m in eval_g!");
    }

    Index i;

    if (TURN_OFF_INPUT_CONSTRAINTS) {
        #pragma omp parallel for shared(kinematics_dynamics_result, x, link_sliced_center) private(i) schedule(dynamic)
        for(i = 0; i < NUM_TIME_STEPS; i++) {
            for (int l = 0; l < NUM_JOINTS; l++) {
                MatrixXInt res = kinematics_dynamics_result->links(l, i).slice(x);
                link_sliced_center[i * NUM_JOINTS + l] = getCenter(res);
            }
        }

        obstacles->linkFRSConstraints(link_sliced_center, nullptr, g, nullptr);

        desired_trajectory->returnJointPositionExtremum(g + NUM_TIME_STEPS * NUM_JOINTS * obstacles->num_obstacles, x);
        desired_trajectory->returnJointVelocityExtremum(g + NUM_TIME_STEPS * NUM_JOINTS * obstacles->num_obstacles + NUM_FACTORS * 2, x);
    }
    else {
        #pragma omp parallel for shared(kinematics_dynamics_result, x, g, link_sliced_center) private(i) schedule(dynamic)
        for(i = 0; i < NUM_TIME_STEPS; i++) {
            for (int k = 0; k < NUM_FACTORS; k++) {
                MatrixXInt res = kinematics_dynamics_result->u_nom(k, i).slice(x);
                g[i * NUM_FACTORS + k] = getCenter(res(0));
            }

            for (int l = 0; l < NUM_JOINTS; l++) {
                MatrixXInt res = kinematics_dynamics_result->links(l, i).slice(x);
                link_sliced_center[i * NUM_JOINTS + l] = getCenter(res);
            }
        }

        obstacles->linkFRSConstraints(link_sliced_center, nullptr, g + NUM_TIME_STEPS * NUM_FACTORS, nullptr);

        desired_trajectory->returnJointPositionExtremum(g + NUM_TIME_STEPS * NUM_FACTORS + NUM_TIME_STEPS * NUM_JOINTS * obstacles->num_obstacles, x);
        desired_trajectory->returnJointVelocityExtremum(g + NUM_TIME_STEPS * NUM_FACTORS + NUM_TIME_STEPS * NUM_JOINTS * obstacles->num_obstacles + NUM_FACTORS * 2, x);
    }

    return true;
}
// [TNLP_eval_g]


// [TNLP_eval_jac_g]
// return the structure or values of the Jacobian
bool armtd_NLP::eval_jac_g(
   Index         n,
   const Number* x,
   bool          new_x,
   Index         m,
   Index         nele_jac,
   Index*        iRow,
   Index*        jCol,
   Number*       values
)
{
    if(n != NUM_FACTORS){
        WARNING_PRINT("*** Error wrong value of n in eval_g!");
    }
    if(m != constraint_number){
        WARNING_PRINT("*** Error wrong value of m in eval_g!");
    }
        
    if( values == NULL ) {
       // return the structure of the Jacobian
       // this particular Jacobian is dense
        for(Index i = 0; i < m; i++){
            for(Index j = 0; j < n; j++){
                iRow[i * n + j] = i;
                jCol[i * n + j] = j;
            }
        }
    }
    else {
        Index i;

        if (TURN_OFF_INPUT_CONSTRAINTS) {
            #pragma omp parallel for shared(kinematics_dynamics_result, x, link_sliced_center, dk_link_sliced_center) private(i) schedule(dynamic)
            for(i = 0; i < NUM_TIME_STEPS; i++) {
                for (int l = 0; l < NUM_JOINTS; l++) {
                    link_sliced_center[i * NUM_JOINTS + l] = getCenter(kinematics_dynamics_result->links(l, i).slice(x));
                    kinematics_dynamics_result->links(l, i).slice(dk_link_sliced_center + (i * NUM_JOINTS + l) * NUM_FACTORS, x);
                }
            }

            obstacles->linkFRSConstraints(link_sliced_center, dk_link_sliced_center, nullptr, values);

            desired_trajectory->returnJointPositionExtremumGradient(values + (NUM_TIME_STEPS * NUM_JOINTS * obstacles->num_obstacles) * NUM_FACTORS, x);
            desired_trajectory->returnJointVelocityExtremumGradient(values + (NUM_TIME_STEPS * NUM_JOINTS * obstacles->num_obstacles + NUM_FACTORS * 2) * NUM_FACTORS, x);
        }
        else {
            #pragma omp parallel for shared(kinematics_dynamics_result, x, values, link_sliced_center, dk_link_sliced_center) private(i) schedule(dynamic)
            for(i = 0; i < NUM_TIME_STEPS; i++) {
                for (int k = 0; k < NUM_FACTORS; k++) {
                    kinematics_dynamics_result->u_nom(k, i).slice(values + (i * NUM_FACTORS + k) * NUM_FACTORS, x);
                }

                for (int l = 0; l < NUM_JOINTS; l++) {
                    link_sliced_center[i * NUM_JOINTS + l] = getCenter(kinematics_dynamics_result->links(l, i).slice(x));
                    kinematics_dynamics_result->links(l, i).slice(dk_link_sliced_center + (i * NUM_JOINTS + l) * NUM_FACTORS, x);
                }
            }

            obstacles->linkFRSConstraints(link_sliced_center, dk_link_sliced_center, nullptr, values + NUM_TIME_STEPS * NUM_FACTORS * NUM_FACTORS);

            desired_trajectory->returnJointPositionExtremumGradient(values + (NUM_TIME_STEPS * NUM_FACTORS + NUM_TIME_STEPS * NUM_JOINTS * obstacles->num_obstacles) * NUM_FACTORS, x);
            desired_trajectory->returnJointVelocityExtremumGradient(values + (NUM_TIME_STEPS * NUM_FACTORS + NUM_TIME_STEPS * NUM_JOINTS * obstacles->num_obstacles + NUM_FACTORS * 2) * NUM_FACTORS, x);
        }
    }

    return true;
}
// [TNLP_eval_jac_g]


// [TNLP_eval_h]
//return the structure or values of the Hessian
bool armtd_NLP::eval_h(
   Index         n,
   const Number* x,
   bool          new_x,
   Number        obj_factor,
   Index         m,
   const Number* lambda,
   bool          new_lambda,
   Index         nele_hess,
   Index*        iRow,
   Index*        jCol,
   Number*       values
)
{
    return false;
}
// [TNLP_eval_h]


// [TNLP_finalize_solution]
void armtd_NLP::finalize_solution(
    SolverReturn               status,
    Index                      n,
    const Number*              x,
    const Number*              z_L,
    const Number*              z_U,
    Index                      m,
    const Number*              g,
    const Number*              lambda,
    Number                     obj_value,
    const IpoptData*           ip_data,
    IpoptCalculatedQuantities* ip_cq
)
{
    // here is where we would store the solution to variables, or write to a file, etc
    // so we could use the solution.

    // store the solution
    for( Index i = 0; i < n; i++ ) {
        solution[i] = (double)x[i];
    }

    cout << "        CUDA & C++: Ipopt: final cost function value: " << obj_value / COST_FUNCTION_OPTIMALITY_SCALE << endl;

    // check constraint violation manually for Maximum_CpuTime_Exceeded case
    memcpy(g_copy, g, m * sizeof(Number));

    feasible = true;

    Index offset = 0;

    if (!TURN_OFF_INPUT_CONSTRAINTS) {
        // control input constraints
        for( Index i = 0; i < NUM_TIME_STEPS; i++ ) {
            for( Index j = 0; j < NUM_FACTORS; j++ ) {
                if (g_copy[i * NUM_FACTORS + j] < -torque_limits[j] + (*torque_radius)(j, i) - TORQUE_INPUT_CONSTRAINT_VIOLATION_THRESHOLD || 
                    g_copy[i * NUM_FACTORS + j] > torque_limits[j] - (*torque_radius)(j, i) + TORQUE_INPUT_CONSTRAINT_VIOLATION_THRESHOLD) {
                    feasible = false;
                    cout << "        CUDA & C++: Ipopt: Control torque of joint " << j << " at time interval " << i << " exceeds limit!\n";
                    cout << "                        value: " << g_copy[i * NUM_FACTORS + j] << "\n";
                    cout << "                        range: [ " << -torque_limits[j] + (*torque_radius)(j, i) << ", "
                                                                << torque_limits[j] - (*torque_radius)(j, i) << " ]\n";
                    return;
                }
            }
        }    

        offset += NUM_FACTORS * NUM_TIME_STEPS;
    }

    // collision avoidance constraints
    for( Index i = 0; i < NUM_JOINTS; i++ ) {
        for( Index j = 0; j < NUM_TIME_STEPS; j++ ) {
            for( Index h = 0; h < obstacles->num_obstacles; h++ ) {
                if (g_copy[(i * NUM_TIME_STEPS + j) * obstacles->num_obstacles + h + offset] > COLLISION_AVOIDANCE_CONSTRAINT_VIOLATION_THRESHOLD) {
                    feasible = false;
                    cout << "        CUDA & C++: Ipopt: Collision between link " << i + 1 << " and obstacle " << h << " at time interval " << j << "!\n";
                    cout << "                        value: " << g_copy[(i * NUM_TIME_STEPS + j) * obstacles->num_obstacles + h + offset] << "\n";
                    return;
                }
            }
        }
    }
    offset += NUM_JOINTS * NUM_TIME_STEPS * obstacles->num_obstacles;

    // state limit constraints
    //     minimum joint position
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        if (g_copy[i] < state_limits_lb[i - offset] + qe || g_copy[i] > state_limits_ub[i - offset] - qe) {
            feasible = false;
            cout << "        CUDA & C++: Ipopt: joint " << i - offset << " exceeds position limit when it reaches minimum!\n";
            cout << "                        value: " << g_copy[i] << "\n";
            cout << "                        range: [ " << state_limits_lb[i - offset] + qe << ", "
                                                        << state_limits_ub[i - offset] - qe << " ]\n";
            return;
        }
    }
    offset += NUM_FACTORS;

    //     maximum joint position
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        if (g_copy[i] < state_limits_lb[i - offset] + qe || g_copy[i] > state_limits_ub[i - offset] - qe) {
            feasible = false;
            cout << "        CUDA & C++: Ipopt: joint " << i - offset << " exceeds position limit when it reaches maximum!\n";
            cout << "                        value: " << g_copy[i] << "\n";
            cout << "                        range: [ " << state_limits_lb[i - offset] + qe << ", "
                                                        << state_limits_ub[i - offset] - qe << " ]\n";
            return;
        }
    }
    offset += NUM_FACTORS;

    //     minimum joint velocity
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        if (g_copy[i] < -speed_limits[i - offset] + qde || g_copy[i] > speed_limits[i - offset] - qde) {
            feasible = false;
            cout << "        CUDA & C++: Ipopt: joint " << i - offset << " exceeds velocity limit when it reaches minimum!\n";
            cout << "                        value: " << g_copy[i] << "\n";
            cout << "                        range: [ " << -speed_limits[i - offset] + qde << ", "
                                                        << speed_limits[i - offset] - qde << " ]\n";
            return;
        }
    }
    offset += NUM_FACTORS;

    //     maximum joint velocity
    for( Index i = offset; i < offset + NUM_FACTORS; i++ ) {
        if (g_copy[i] < -speed_limits[i - offset] + qde || g_copy[i] > speed_limits[i - offset] - qde) {
            feasible = false;
            cout << "        CUDA & C++: Ipopt: joint " << i - offset << " exceeds velocity limit when it reaches maximum!\n";
            cout << "                        value: " << g_copy[i] << "\n";
            cout << "                        range: [ " << -speed_limits[i - offset] + qde << ", "
                                                        << speed_limits[i - offset] - qde << " ]\n";
            return;
        }
    }
}
// [TNLP_finalize_solution]


#endif
